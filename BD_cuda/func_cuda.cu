#include "hip/hip_runtime.h"
#include "func_cuda.h"

double* d_D;
double* d_Chol;

//Device Matrices
int* d_type;
double* d_r;
double* d_a;
double* d_b;
double* d_c;
double* d_alpha;
double* d_beta;
double* d_gamma;
double* d_length;
double* d_a_dna;
double* d_b_dna;
double* d_c_dna;
double* d_alpha_p;
double* d_beta_p;
double* d_gamma_p;
double* d_phi_o;
double* d_force;
double* d_torque;
double* d_Energy;
double* d_core_pos;
double* d_core_q;
double* d_d_theta;
double* d_rd;

double* d_r_n;
double* d_a_n;
double* d_b_n;
double* d_c_n;
double* d_alpha_n;
double* d_beta_n;
double* d_gamma_n;
double* d_length_n;
double* d_a_dna_n;
double* d_b_dna_n;
double* d_c_dna_n;
double* d_alpha_p_n;
double* d_beta_p_n;
double* d_gamma_p_n;
double* d_force_n;
double* d_torque_n;

double* d_force_tmp;
double* d_torque_tmp;

double* d_tail_pos;
int* d_tail_fix;
int* d_nc_t_flag;
double* d_r_t;
double* d_beta_t;
double* d_h_t;
double* d_g_t;
double* d_lo_t;
double* d_beta_o_t;
double* d_t_q;
double* d_t_rad;
int* d_t_grp;
int* d_t_fix;
double* d_t_force;

double* d_r_t_n;
double* d_beta_t_n;

double* d_t_force_n;
double* d_t_force_tmp;

double* d_LH_g_pos;
int* d_LH_conn;
int* d_nc_lh_flag;
double* d_beta_lh;
double* d_r_lh;
double* d_LH_q;
double* d_LH_vdw_hh;
double* d_LH_vdw_hc;
double* d_LH_vdw_hl;
double* d_LH_vdw_ht;
double* d_LH_kstr;
double* d_LH_kben;
double* d_LH_streq;
double* d_LH_betaeq;
double* d_LH_force;

double* d_beta_lh_n;
double* d_r_lh_n;

double* d_LH_force_n;
double* d_LH_force_tmp;

double* d_force_m;
double* d_torque_m;
double* d_t_force_m;
double* d_LH_force_m;

double* d_temp_force_m;

double* d_rad_all;
double* d_r_all;

double* d_Energy_m;

__device__ void first_coord_cuda(int t, double* r, double* a, double* b, double* c, double* r_f){

        double b_m[3];
        double si, co;

        si = sin(theta);
        co = cos(theta);

        if (t==0){
                r_f[0] = r[0];
                r_f[1] = r[1];
                r_f[2] = r[2];
        }else{
                b_m[0] = -si*a[0]+co*b[0];
                b_m[1] = -si*a[1]+co*b[1];
                b_m[2] = -si*a[2]+co*b[2];

                r_f[0] = r[0] - ro*b_m[0]+d1*c[0];
                r_f[1] = r[1] - ro*b_m[1]+d1*c[1];
                r_f[2] = r[2] - ro*b_m[2]+d1*c[2];
        }

}

__device__ void second_coord_cuda(int t, double* r, double* a, double* b, double* c, double* r_s){

        if (t==0){
                r_s[0] = r[0];
                r_s[1] = r[1];
                r_s[2] = r[2];
        }else{
                r_s[0] = r[0] - (ro*b[0]+d1*c[0]);
                r_s[1] = r[1] - (ro*b[1]+d1*c[1]);
                r_s[2] = r[2] - (ro*b[2]+d1*c[2]);
        }

}

__device__ void norm_cuda(double* r, double& result){
    int i;
    double norma = 0;
    for (i = 0; i<3; i++){
        norma = norma + r[i]*r[i];
    }
    result = sqrt(norma);
}

__device__ void cross_product_cuda(double* r1, double* r2, double* product){

    product[0] = r1[1] * r2[2] - r1[2] * r2[1];
    product[1] = -1 * (r1[0] * r2[2] - r1[2] * r2[0]);
    product[2] = r1[0] * r2[1] - r1[1] * r2[0];

}

__device__ void rotate_cuda(int n, int n3, double* a, double* b, double* c, double* a_n, double* b_n, double* c_n, double* d_theta, double dt){

        double wa, wb, wc, g1, g2, g3;
        double z, z2, wa2, wb2, wc2, czt, Omczt, szt;
        int i, i1, i2, i3;

        for (i = 0; i < n; i++){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                wa = d_theta[i1];
                wb = d_theta[i2];
                wc = d_theta[i3];
                wa2 = wa*wa;
                wb2 = wb*wb;
                wc2 = wc*wc;
                z2 = wa2 + wb2 + wc2;
                z = sqrt(z2);
                czt = cos(z*dt);
                szt = sin(z*dt);
                Omczt = 1.0 - czt;

                if (z2 > 0.0){
                        //rotation of a
                        g1 = ((wb2+wc2)*czt+wa2)/z2;
                        g2 = wa*wb*Omczt/z2 + wc*szt/z;
                        g3 = wa*wc*Omczt/z2 - wb*szt/z;
                        a_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        a_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        a_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                        //rotation of b
                        g1 = wa*wb*Omczt/z2 - wc*szt/z;
                        g2 = ((wa2+wc2)*czt+wb2)/z2;
                        g3 = wb*wc*Omczt/z2 + wa*szt/z;
                        b_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        b_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        b_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                        //rotation of c
                        g1 = wa*wc*Omczt/z2 + wb*szt/z;
                        g2 = wb*wc*Omczt/z2 - wa*szt/z;
                        g3 = ((wa2+wb2)*czt+wc2)/z2;
                        c_n[i1] = g1*a[i1] + g2*b[i1] + g3*c[i1];
                        c_n[i2] = g1*a[i2] + g2*b[i2] + g3*c[i2];
                        c_n[i3] = g1*a[i3] + g2*b[i3] + g3*c[i3];

                }else{
                        a_n[i1] = a[i1];
                        a_n[i2] = a[i2];
                        a_n[i3] = a[i3];
                        b_n[i1] = b[i1];
                        b_n[i2] = b[i2];
                        b_n[i3] = b[i3];
                        c_n[i1] = c[i1];
                        c_n[i2] = c[i2];
                        c_n[i3] = c[i3];
                }

        }

}

__device__ void Force_Stretching_cuda(double h, double* coord1, double* coord2, double lo, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;

        double distance = 0.0;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

	distance = sqrt(distance);

        Energy = Energy + h*(distance-lo)*(distance-lo)/2;

        force = -h*(distance-lo);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
        }

}

__device__ void Force_Bending_cuda(double g, double beta, double beta_o, double* coord1, double* coord2, double* coord3, double* force_projection1, double* force_projection2, double* force_projection3, double& Energy){

        double force = 0.0;

        double distance1, distance2;
        double norm_ri, norm_rk;

        double ji[3], jk[3], kj[3], ri[3], rk[3], product[3];

        Energy = Energy + g*(beta-beta_o)*(beta-beta_o)/2;

        force = -g*(beta-beta_o);

        distance1 = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);
        distance2 = (coord2[0]-coord3[0])*(coord2[0]-coord3[0]) + (coord2[1]-coord3[1])*(coord2[1]-coord3[1]) + (coord2[2]-coord3[2])*(coord2[2]-coord3[2]);

	distance1 = sqrt(distance1);
	distance2 = sqrt(distance2);

	ji[0] = coord2[0]-coord1[0];
	ji[1] = coord2[1]-coord1[1];
	ji[2] = coord2[2]-coord1[2];
	jk[0] = coord2[0]-coord3[0];
	jk[1] = coord2[1]-coord3[1];
	jk[2] = coord2[2]-coord3[2];
	kj[0] = coord3[0]-coord2[0];
	kj[1] = coord3[1]-coord2[1];
	kj[2] = coord3[2]-coord2[2];
        
	cross_product_cuda(ji, jk, product);
	for (int i = 0; i < 3; i++){
		ri[i] = product[i];
	}
        cross_product_cuda(ji, ri, product);
	for (int i = 0; i < 3; i++){
                ri[i] = product[i];
        }

        norm_cuda(ri, norm_ri);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = (force/distance1)*ri[i]/norm_ri;
                force_projection2[i] = -(force/distance1)*ri[i]/norm_ri;
        }

        cross_product_cuda(ji, jk, product);
	for (int i = 0; i < 3; i++){
                rk[i] = product[i];
        }
        cross_product_cuda(kj, rk, product);
	for (int i = 0; i < 3; i++){
                rk[i] = product[i];
        }

        norm_cuda(rk, norm_rk);

        for (int j = 0; j < 3; j++){
                force_projection2[j] = force_projection2[j] - (force/distance2)*rk[j]/norm_rk;
                force_projection3[j] = (force/distance2)*rk[j]/norm_rk;
        }

}


__device__ void Bending_force_projection_cuda(double g, double beta, double beta_b, double length, double* a_f, double* a_b, double* a, double* force_projection1, double* force_projection2, double& Energy){

        double Ai[3], Bi[3];
        double c1, c2, g1, g2;

        Energy = Energy + g*(beta-beta_b)*(beta-beta_b)/2;

        if (beta >= 1e-10){
                g1 = beta/(sin(beta)*length);
        }else{
                g1 = 1.0/length;
        }
        c1 = cos(beta);

        Ai[0] = g1*(a_f[0]-c1*a[0]);
        Ai[1] = g1*(a_f[1]-c1*a[1]);
        Ai[2] = g1*(a_f[2]-c1*a[2]);

        if (beta_b >= 1e-10){
                g2 = beta_b/(sin(beta_b)*length);
        }else{
                g2 = 1.0/length;
        }
        c2 = cos(beta_b);

        Bi[0] = g2*(a_b[0]-c2*a[0]);
        Bi[1] = g2*(a_b[1]-c2*a[1]);
        Bi[2] = g2*(a_b[2]-c2*a[2]);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = -g*(Ai[i]+Bi[i]);
                force_projection2[i] = g*(Ai[i]+Bi[i]);
        }

}

__device__ void Twisting_force_projection_cuda(double s, double alpha, double beta, double gamma, double phi_o, double length, double alpha_b, double beta_b, double gamma_b, double phi_o_b, double gamma_n, double* b, double* c, double* force_projection1, double* force_projection2, double& Energy){

        double Chi[3], Zhi[3];
        double g1, g2, c1, c2, s1, s2;

        Energy = Energy + s*((alpha+gamma-phi_o)*(alpha+gamma-phi_o))/2;

        g1 = (alpha+gamma-phi_o)*tan(0.5*beta)/length;
        c1 = cos(alpha);
        s1 = sin(alpha);

        g2 = (alpha_b+gamma_b-phi_o_b)*tan(0.5*beta_b)/length;
        c2 = cos(gamma_n);
        s2 = sin(gamma_n);

        Chi[0] = g1*(c1*c[0]-s1*b[0]);
        Chi[1] = g1*(c1*c[1]-s1*b[1]);
        Chi[2] = g1*(c1*c[2]-s1*b[2]);
        Zhi[0] = g2*(c2*c[0]+s2*b[0]);
        Zhi[1] = g2*(c2*c[1]+s2*b[1]);
        Zhi[2] = g2*(c2*c[2]+s2*b[2]);

        for (int i = 0; i < 3; i++){
                force_projection1[i] = s*(Chi[i]+Zhi[i]);
                force_projection2[i] = -s*(Chi[i]+Zhi[i]);
        }
}

__device__ void Force_Electrostatics_cuda(double q1, double q2, double epslon, double kappa, double* coord1, double* coord2, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;
        double Rcut = 7.0;
        double distance = 0.0;
	double temp;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

	distance = sqrt(distance);

        if (distance < Rcut){
		temp = -kappa*distance;
                force = ((q1*q2*(kappa*distance+1))/(4*PI*epslon*distance*distance))*exp(temp);
                Energy = Energy + (q1*q2/(4*PI*epslon*distance))*exp(temp);
        }

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
                if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void Force_Exclude_Volume_cuda(double k_ev, double sigma, double* coord1, double* coord2, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;
        double vdw_cut = 4.0;
        double distance = 0.0;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

	distance = sqrt(distance);

        if (distance < vdw_cut){
                force = -k_ev*((6*pow(sigma,6))/pow(distance,7) - (12*pow(sigma,12))/pow(distance,13));
                Energy = Energy + k_ev*(pow(sigma/distance,12) - pow(sigma/distance,6));
        }

        for (int i = 0; i < 3; i++){
                force_projection1[i] = force*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -force*(coord1[i] - coord2[i])/distance;
                if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void Force_Ele_Vdw_cuda(double q1, double q2, double epslon, double kappa, double k_ev, double sigma, double* coord1, double* coord2, double* force_projection1, double* force_projection2, double& Energy){

        double force = 0.0;
        double force2 = 0.0;
        double Rcut = 7.0;
        double vdw_cut = 4.0;
        double distance = 0.0;
	double temp;

        distance = (coord1[0]-coord2[0])*(coord1[0]-coord2[0]) + (coord1[1]-coord2[1])*(coord1[1]-coord2[1]) + (coord1[2]-coord2[2])*(coord1[2]-coord2[2]);

	distance = sqrt(distance);

        if (distance < Rcut){
		temp = -kappa*distance;
                force = ((q1*q2*(kappa*distance+1))/(4*PI*epslon*distance*distance))*exp(temp);
                Energy = Energy + (q1*q2/(4*PI*epslon*distance))*exp(temp);
        }

        if (distance < vdw_cut){
                force2 = -k_ev*((6*pow(sigma,6))/pow(distance,7) - (12*pow(sigma,12))/pow(distance,13));
                Energy = Energy + k_ev*(pow(sigma/distance,12) - pow(sigma/distance,6));
        }

        for (int i = 0; i < 3; i++){
                force_projection1[i] = (force+force2)*(coord1[i] - coord2[i])/distance;
                force_projection2[i] = -(force+force2)*(coord1[i] - coord2[i])/distance;
                if (force_projection1[i]>0.9){ force_projection1[i]=0.9; }
                if (force_projection2[i]>0.9){ force_projection2[i]=0.9; }
                if (force_projection1[i]<-0.9){ force_projection1[i]=-0.9; }
                if (force_projection2[i]<-0.9){ force_projection2[i]=-0.9; }
        }

}

__device__ void torque_due_to_force_cuda(double* force, double* coord_f, double* coord_c, double* a, double* b, double* c, double* torque){

        //Given exact coordinates of the bead with force applied (coord_f) and the exact coordinate of the center bead (coord_c)

        double fa, fb, fc;
        double comp[3];

        comp[0] = (coord_f[0]-coord_c[0])*a[0] + (coord_f[1]-coord_c[1])*a[1] + (coord_f[2]-coord_c[2])*a[2];
        comp[1] = (coord_f[0]-coord_c[0])*b[0] + (coord_f[1]-coord_c[1])*b[1] + (coord_f[2]-coord_c[2])*b[2];
        comp[2] = (coord_f[0]-coord_c[0])*c[0] + (coord_f[1]-coord_c[1])*c[1] + (coord_f[2]-coord_c[2])*c[2];


        fa = a[0]*force[0] + a[1]*force[1] + a[2]*force[2];
        fb = b[0]*force[0] + b[1]*force[1] + b[2]*force[2];
        fc = c[0]*force[0] + c[1]*force[1] + c[2]*force[2];

        torque[0] = fc*comp[1] - fb*comp[2];
        torque[1] = fa*comp[2] - fc*comp[0];
        torque[2] = fb*comp[0] - fa*comp[1];

}

__device__ void torque_due_to_force_relative_cuda(double* force, double* comp, double* a, double* b, double* c, double* torque){

        //Given relative position (comp) of the bead with force applied

        double fa, fb, fc;

        fa = a[0]*force[0] + a[1]*force[1] + a[2]*force[2];
        fb = b[0]*force[0] + b[1]*force[1] + b[2]*force[2];
        fc = c[0]*force[0] + c[1]*force[1] + c[2]*force[2];

        torque[0] = fc*comp[1] - fb*comp[2];
        torque[1] = fa*comp[2] - fc*comp[0];
        torque[2] = fb*comp[0] - fa*comp[1];

}

__global__ void Diffusion_Tensor_CUDA(int n, int n3, double* r, double a1, double a2, double* rad, double* D, double* Chol){

	// Calculate Row and Column
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	// indexing

	double sij[3];
        double ssq, s, f, f1, f2;
        int i, j, k, l;

	// Diffusion Tensor

	i = row / 3;
	j = column / 3;
	k = row % 3;
	l = column % 3;

	if (row < n3 and column < n3){

		if (row == column){
			D[row*n3+column] = a1/rad[i];
			Chol[row*n3+column] = a1/rad[i];
		} else if (i==j and k!=l){
			D[row*n3+column] = 0;
			Chol[row*n3+column] = 0;
		} else {
			sij[0] = r[3*i] - r[3*j];
	                sij[1] = r[3*i+1] - r[3*j+1];
        	        sij[2] = r[3*i+2] - r[3*j+2];
                	ssq = sij[0]*sij[0] + sij[1]*sij[1] + sij[2]*sij[2];
	                s = sqrt(ssq);
			
			if (s >= rad[i]+rad[j]){
				f = (rad[i]*rad[i]+rad[j]*rad[j])/ssq;
				f1 = 1.0 + f/3;
                        	f2 = 1.0 -f;
				if (k==l){
					D[row*n3+column] = (a2/s)*(f1+f2*sij[k]*sij[k]/ssq);
					Chol[row*n3+column] = (a2/s)*(f1+f2*sij[k]*sij[k]/ssq);
				} else {
					D[row*n3+column] = (a2/s)*(f2*sij[k]*sij[l]/ssq);
					Chol[row*n3+column] = (a2/s)*(f2*sij[k]*sij[l]/ssq);
				}

			}else{
				s = pow((rad[i]*rad[i]*rad[i]+rad[j]*rad[j]*rad[j])/2.0, 1.0/3);
				if (k==l){
					D[row*n3+column] = (a1/s)*(1.0-9.0*sqrt(ssq)/(32*s) + 3.0*sij[k]*sij[k]/(32.0*s*sqrt(ssq)));
					Chol[row*n3+column] = (a1/s)*(1.0-9.0*sqrt(ssq)/(32*s) + 3.0*sij[k]*sij[k]/(32.0*s*sqrt(ssq)));
				} else {
					D[row*n3+column] = (a1/s)*(3.0*sij[k]*sij[l]/(32.0*s*sqrt(ssq)));
					Chol[row*n3+column] = (a1/s)*(3.0*sij[k]*sij[l]/(32.0*s*sqrt(ssq)));
				}
			}
		}

	}
}

__global__ void Cholesky_Decomposition_CUDA(double* D, int n3, double* Chol){

	// Calculate Row and Column
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int column = blockIdx.x * blockDim.x + threadIdx.x;

	// indexing

	double ssq, f;
	int i,j,k;

	//Cholesky decomposition

	if (row < n3 and column < n3){

		Chol[row*n3+column] = 0;
		
                if (row == 0 and column == 0){
                        Chol[0] = sqrt(D[0]);
                
                	for (i=1; i<n3; i++){
                                
                                Chol[i*n3] = D[i*n3]/Chol[0];
                        
                                for (j=1;j<i;j++){
                                        ssq = 0.0;
                                        for (k=0;k<j;k++){
                                                ssq += Chol[i*n3+k]*Chol[j*n3+k];
                                        }
                                      
                                        Chol[i*n3+j] = (D[i*n3+j]-ssq)/Chol[j*n3+j];
                                }

                                ssq = 0.0;

                                for (k=0;k<i;k++){
                                        ssq += Chol[i*n3+k]*Chol[i*n3+k];
                                }
                                
                                f = D[i*n3+i] - ssq;

                                if (f < 0){
                                        f = -f;
                                }
                                Chol[i*n3+i] = sqrt(f);
                        }
                }

	}


}

__global__ void Cholesky_Decomposition_mod(int n3, double* Chol){

        // Calculate Row and Column
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int column = blockIdx.x * blockDim.x + threadIdx.x;


        if (row < n3 and column < n3){
                if (row < column){
                        Chol[row*n3+column] = 0.0;
                }
		if (isnan(Chol[row*n3+column])){
			Chol[row*n3+column] = 0.0;
		}
        }

}



__global__ void rd_cal(int n3, double* rd, double* Chol, double* p, double s2dt){

	int column = blockIdx.x * blockDim.x + threadIdx.x;

	if (column < n3){

		rd[column] = 0.0;
	
		for (int k = 0; k <= column; k++){
			rd[column] += s2dt*Chol[column*n3+k]*p[k];
		}

	}
}


__global__ void translation_cal(int n_D3, int n3, int n_tail3, int n_LH3, double* r, double* r_n, double* rd, double* r_t, double* r_t_n, double* r_lh, double* r_lh_n, double del, double* force_global, double* t_force_global, double* LH_force_global, double* D){

	int column = blockIdx.x * blockDim.x + threadIdx.x;

	int k;

	if (column < n_D3){

	
                if (column < n3){
                        r_n[column] = r[column] + rd[column];
                        for (k = 0; k < n3; k++){
                                r_n[column] = r_n[column] + del*D[column*n_D3+k]*force_global[k];
			}
                }else if (column < n3+n_tail3){
                        r_t_n[column-n3] = r_t[column-n3] + rd[column];
                        for (k = n3; k < n3+n_tail3; k++){
                                r_t_n[column-n3] = r_t_n[column-n3] + del*D[column*n_D3+k]*t_force_global[k-n3];
                        }
                }else{
                        r_lh_n[column-n3-n_tail3] = r_lh[column-n3-n_tail3] + rd[column];
                        for (k = n3+n_tail3; k < n3+n_tail3+n_LH3; k++){
                                r_lh_n[column-n3-n_tail3] = r_lh_n[column-n3-n_tail3] + del*D[column*n_D3+k]*LH_force_global[k-n3-n_tail3];
                        }
                }
        

	}

}

__global__ void rotation_cal(int n, double* d_theta, int* type, double time_step, double* torque, double* rr, double* a, double* b, double* c, double* a_n, double* b_n, double* c_n){

	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int j1, j2, j3;

	if (j < n){
		j1 = 3*j;
                j2 = j1+1;
                j3 = j2+1;
                if (type[j] != 0){
                	d_theta[j1] = time_step*torque[j1]/(8*PI*eta*125.0) + rr[j1];
                        d_theta[j2] = time_step*torque[j2]/(8*PI*eta*125.0) + rr[j2];
                        d_theta[j3] = time_step*torque[j3]/(8*PI*eta*125.0) + rr[j3];
                }else{
                        d_theta[j1] = time_step*torque[j1]/(4*PI*eta*r_h*r_h*lo) + rr[j1];
                        d_theta[j2] = 0.0;
                        d_theta[j3] = 0.0;
                }	
	}

	rotate_cuda(n, n*3, a, b, c, a_n, b_n, c_n, d_theta, 1.0);

}

__global__ void update_Euler_Angle_cuda(int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        double r_forw[3];
        double mi;
        double da[3], a_old[3];
        double a_m[3], b_m[3];
        double Ac, apg, f1, f2, ada, bda, si, co;
        double sa, ca, sb, cb, sg, cg;
        double R21, R22, R23, R31, R32, R33;
        int i1,i2,i3, if1,if2,if3, ic,ic1,ic2,ic3;
        int count;

        si = sin(theta);
        co = cos(theta);

        if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                if1 = i1+3;
                if2 = if1+1;
                if3 = if2+1;

                if (type[i] == 0){
                        a_old[0] = a[i1];
                        a_old[1] = a[i2];
                        a_old[2] = a[i3];
                        if (type[i+1] == 0){
                                r_forw[0] = r[if1] - r[i1];
                                r_forw[1] = r[if2] - r[i2];
                                r_forw[2] = r[if3] - r[i3];
                        }else{
                                b_m[0] = -si*a[if1] + co*b[if1];
                                b_m[1] = -si*a[if2] + co*b[if2];
                                b_m[2] = -si*a[if3] + co*b[if3];
                                r_forw[0] = r[if1] - ro*b_m[0] + d1*c[if1] - r[i1];
                                r_forw[1] = r[if2] - ro*b_m[1] + d1*c[if2] - r[i2];
                                r_forw[2] = r[if3] - ro*b_m[2] + d1*c[if3] - r[i3];
                        }
                        length[i] = r_forw[0]*r_forw[0] + r_forw[1]*r_forw[1] + r_forw[2]*r_forw[2];
			length[i] = sqrt(length[i]);
                        mi = 1.0/length[i];
                        a[i1] = mi*r_forw[0];
                        a[i2] = mi*r_forw[1];
                        a[i3] = mi*r_forw[2];
                        da[0] = a[i1] - a_old[0];
                        da[1] = a[i2] - a_old[1];
                        da[2] = a[i3] - a_old[2];

                        bda = b[i1]*da[0] +b[i2]*da[1] + b[i3]*da[2];
                        b[i1] = b[i1] - bda*a_old[0];
                        b[i2] = b[i2] - bda*a_old[1];
                        b[i3] = b[i3] - bda*a_old[2];

                        bda = b[i1]*a[i1] + b[i2]*a[i2] + b[i3]*a[i3];
                        b[i1] = b[i1] - bda*a[i1];
                        b[i2] = b[i2] - bda*a[i2];
                        b[i3] = b[i3] - bda*a[i3];

                        mi = 1.0/sqrt(b[i1]*b[i1] + b[i2]*b[i2] + b[i3]*b[i3]);
                        b[i1] = mi*b[i1];
                        b[i2] = mi*b[i2];
                        b[i3] = mi*b[i3];

                        c[i1] = a[i2]*b[i3] - a[i3]*b[i2];
                        c[i2] = a[i3]*b[i1] - a[i1]*b[i3];
                        c[i3] = a[i1]*b[i2] - a[i2]*b[i1];
                }else{
                        r_forw[0] = r[if1] - (r[i1]-ro*b[i1]-d1*c[i1]);
                        r_forw[1] = r[if2] - (r[i2]-ro*b[i2]-d1*c[i2]);
                        r_forw[2] = r[if3] - (r[i3]-ro*b[i3]-d1*c[i3]);
                        length[i] = r_forw[0]*r_forw[0] + r_forw[1]*r_forw[1] + r_forw[2]*r_forw[2];
			length[i] = sqrt(length[i]);
                }
        }

	if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;
                if1 = i1+3;
                if2 = if1+1;
                if3 = if2+1;

                if (type[i]==0){
                        if(type[i+1]==0){
                                ada = a[i1]*a[if1]+a[i2]*a[if2]+a[i3]*a[if3];
                                if (ada > 1.0) ada = 1.0;
                                if (ada < -1.0) ada = -1.0;
                                beta[i] = acos(ada);
                                sb = sin(beta[i]);
                                if (beta[i] > 1e-10){
                                        f1 = (a[if1]*b[i1]+a[if2]*b[i2]+a[if3]*b[i3])/sb;
                                }else{
                                        f1 = (b[if1]*b[i1]+b[if2]*b[i2]+b[if3]*b[i3]);
                                }
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                Ac = acos(f1);
                                f2 = a[if1]*c[i1] + a[if2]*c[i2] + a[if3]*c[i3];
                                if (f2 >= 0){
                                        alpha[i] = Ac;
                                }else{
                                        alpha[i] = -Ac;
                                }

                                f1 = (b[i1]*b[if1]+b[i2]*b[if2]+b[i3]*b[if3]+c[i1]*c[if1]+c[i2]*c[if2]+c[i3]*c[if3])/(1.0 + ada);
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                apg = acos(f1);
                                f2 = (c[i1]*b[if1]+c[i2]*b[if2]+c[i3]*b[if3]-(b[i1]*c[if1]+b[i2]*c[if2]+b[i3]*c[if3]))/(1.0 + ada);
                                if (f2 >= 0.0){
                                        gamma[i] = apg - alpha[i];
                                }else{
                                        gamma[i] = -apg - alpha[i];
                                }
                        }else{
                                a_m[0] = co*a[if1] + si*b[if1];
                                a_m[1] = co*a[if2] + si*b[if2];
                                a_m[2] = co*a[if3] + si*b[if3];
                                b_m[0] = -si*a[if1] + co*b[if1];
                                b_m[1] = -si*a[if2] + co*b[if2];
                                b_m[2] = -si*a[if3] + co*b[if3];

                                ada = a[i1]*a_m[0]+a[i2]*a_m[1]+a[i3]*a_m[2];
                                if (ada > 1.0) ada = 1.0;
                                if (ada < -1.0) ada = -1.0;
                                beta[i] = acos(ada);
                                sb = sin(beta[i]);
                                if (beta[i] > 1e-10){
                                        f1 = (a_m[0]*b[i1]+a_m[1]*b[i2]+a_m[2]*b[i3])/sb;
                                }else{
                                        f1 = (b_m[0]*b[i1]+b_m[1]*b[i2]+b_m[2]*b[i3]);
                                }
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                Ac = acos(f1);
                                f2 = a_m[0]*c[i1] + a_m[1]*c[i2] + a_m[2]*c[i3];
                                if (f2 >= 0){
                                        alpha[i] = Ac;
                                }else{
                                        alpha[i] = -Ac;
                                }

                                f1 = (b[i1]*b_m[0]+b[i2]*b_m[1]+b[i3]*b_m[2]+c[i1]*c[if1]+c[i2]*c[if2]+c[i3]*c[if3])/(1.0 + ada);
                                if (f1 > 1.0) f1 = 1.0;
                                if (f1 < -1.0) f1 = -1.0;
                                apg = acos(f1);
                                f2 = (c[i1]*b_m[0]+c[i2]*b_m[1]+c[i3]*b_m[2]-(b[i1]*c[if1]+b[i2]*c[if2]+b[i3]*c[if3]))/(1.0 + ada);
                                if (f2 >= 0.0){
                                        gamma[i] = apg - alpha[i];
                                }else{
                                        gamma[i] = -apg - alpha[i];
                                }
                        }
                }else{
                        ic = 0;
                        for (count = 0; count <= i; count++){
                                if(type[count]==1) ic=ic+1;
                        }
                        ic = ic-1;
                        ic1 = 3*ic;
                        ic2 = ic1+1;
                        ic3 = ic2+1;

                        a_dna[ic1] = (r[if1] - (r[i1]-ro*b[i1]-d1*c[i1]))/length[i];
                        a_dna[ic2] = (r[if2] - (r[i2]-ro*b[i2]-d1*c[i2]))/length[i];
                        a_dna[ic3] = (r[if3] - (r[i3]-ro*b[i3]-d1*c[i3]))/length[i];

                        cb = a[i1]*a_dna[ic1] + a[i2]*a_dna[ic2] + a[i3]*a_dna[ic3];
                        if (cb > 1.0) cb = 1.0;
                        if (cb < -1.0) cb = -1.0;
                        beta_p[ic] = acos(cb);
                        sb = sin(beta_p[ic]);
                        if (beta_p[ic] >= 1e-10){
                                b_m[0] = (a_dna[ic1]-cb*a[i1])/sb;
                                b_m[1] = (a_dna[ic2]-cb*a[i2])/sb;
                                b_m[2] = (a_dna[ic3]-cb*a[i3])/sb;
                                ca = b_m[0]*b[i1]+b_m[1]*b[i2]+b_m[2]*b[i3];
                                if (ca > 1.0) ca = 1.0;
                                if (ca < -1.0) ca = -1.0;
                                Ac = acos(ca);
                                f1 = a_dna[ic1]*c[i1]+a_dna[ic2]*c[i2]+a_dna[ic3]*c[i3];
                                if (f1 >= 0){
                                        alpha_p[ic] = Ac;
                                }else{
                                        alpha_p[ic] = -Ac;
                                }
                                gamma_p[ic] = -alpha_p[ic];
                                sa = sin(alpha_p[ic]);
                                sg = sin(gamma_p[ic]);
                                cg = cos(gamma_p[ic]);
                                R21 = -cg*sb;
                                R22 = cg*cb*ca-sg*sa;
                                R23 = cg*cb*sa+sg*ca;

                                b_dna[ic1] = R21*a[i1] + R22*b[i1] + R23*c[i1];
                                b_dna[ic2] = R21*a[i2] + R22*b[i2] + R23*c[i2];
                                b_dna[ic3] = R21*a[i3] + R22*b[i3] + R23*c[i3];

                                R31 = sg*sb;
                                R32 = -sg*cb*ca-cg*sa;
                                R33 = -sg*cb*sa+cg*ca;

                                c_dna[ic1] = R31*a[i1]+R32*b[i1]+R33*c[i1];
                                c_dna[ic2] = R31*a[i2]+R32*b[i2]+R33*c[i2];
                                c_dna[ic3] = R31*a[i3]+R32*b[i3]+R33*c[i3];


                        }else{
                                b_dna[ic1] = b[i1];
                                b_dna[ic2] = b[i2];
                                b_dna[ic3] = b[i3];
                                c_dna[ic1] = c[i1];
                                c_dna[ic2] = c[i2];
                                c_dna[ic3] = c[i3];
                        }
                        ada = a_dna[ic1]*a[if1]+a_dna[ic2]*a[if2]+a_dna[ic3]*a[if3];
                        if (ada > 1.0) ada = 1.0;
                        if (ada < -1.0) ada = -1.0;
                        beta[i] = acos(ada);
                        sb  = sin(beta[i]);
                        if (beta[i] >= 1e-10){
                                f1 = (a[if1]*b_dna[ic1]+a[if2]*b_dna[ic2]+a[if3]*b_dna[ic3])/sb;
                        }else{
                                f1 = (b[if1]*b_dna[ic1]+b[if2]*b_dna[ic2]+b[if3]*b_dna[ic3]);
                        }
                        if (f1 > 1.0) f1 =1.0;
                        if (f1 < -1.0) f1 = -1.0;
                        Ac = acos(f1);
                        f2 = a[if1]*c_dna[ic1]+a[if2]*c_dna[ic2]+a[if3]*c_dna[ic3];
                        if (f2 >= 0){
                                alpha[i] = Ac;
                        }else{
                                alpha[i] = -Ac;
                        }

                        f1 = (b_dna[ic1]*b[if1]+b_dna[ic2]*b[if2]+b_dna[ic3]*b[if3]+c_dna[ic1]*c[if1]+c_dna[ic2]*c[if2]+c_dna[ic3]*c[if3])/(1.0+ada);
                        if (f1 > 1.0) f1 = 1.0;
                        if (f1 < -1.0) f1 = -1.0;
                        apg = acos(f1);
                        f2 = (c_dna[ic1]*b[if1]+c_dna[ic2]*b[if2]+c_dna[ic3]*b[if3]-(b_dna[ic1]*c[if1]+b_dna[ic2]*c[if2]+b_dna[ic3]*c[if3]))/(1.0+ada);
                        if (f2 >= 0.0){
                                gamma[i] = apg - alpha[i];
                        }else{
                                gamma[i] = -apg - alpha[i];
                        }

                }

        }



}



__global__ void mechanical_force_and_torque_cuda(int maxrtlh, int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p, double h, double g, double s, double* phi_o, double* force, double* torque, double* Energy, double* force_m, double* torque_m, double* Energy_m){

	int ic, count;
        int i1, i2, i3, ib1, ib2, ib3, ic1, ic2, ic3, im1;
        double c1, s1, si, co;
        double a_m[3];
        double alpha_b, beta_b, gamma_b, phi_o_b, gamma_n;
        double a_f[3], a_b[3], a_o[3], b_o[3], c_o[3];
        double mag;
        double ada, adb, adc, cda, bda, cdb, bdb, cdc, bdc;

        double torque1[3], torque2[3]; 
        double force_projection1[3], force_projection2[3]; 
        double r_f[3], r_s[3]; 

        double r_tmp1[3], r_tmp2[3], a_tmp1[3], a_tmp2[3], b_tmp1[3], b_tmp2[3], c_tmp1[3], c_tmp2[3];


        si = sin(theta);
        co = cos(theta);

	int column = blockIdx.x * blockDim.x + threadIdx.x;

	//Stretching

        if (column<n-1){

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

                first_coord_cuda (type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
		second_coord_cuda (type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                Force_Stretching_cuda (h,r_s,r_f,lo,force_projection1,force_projection2, Energy_m[column]);

                force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

		force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda (force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
                torque_due_to_force_cuda (force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

                torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

		torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

	__threadfence();

	//Bending
        if (column < n-1){

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

                first_coord_cuda(type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
                second_coord_cuda(type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                if (type[column]==0){
                        if (type[column+1]==0){
                                for (int xi = 0; xi<3; xi++){
                                        a_f[xi] = a[column*3+3+xi];
                                }
                        }else{
                                a_m[0] = co*a[column*3+3] + si*b[column*3+3];
                                a_m[1] = co*a[column*3+4] + si*b[column*3+4];
                                a_m[2] = co*a[column*3+5] + si*b[column*3+5];
                                for (int xi = 0; xi<3; xi++){
                                        a_f[xi] = a_m[xi];
                                }
                        }
                        if (type[column-1]==0){
                                for (int xi = 0; xi<3; xi++){
                                        a_b[xi] = a[column*3-3+xi];
                                }
                        }else{
                                for (int xi = 0; xi<3; xi++){
                                        a_b[xi] = a_dna[ic1+xi];
                                }
                        }
                        for (int xi = 0; xi<3; xi++){
                                a_o[xi] = a[column*3+xi];
                        }
                        beta_b = beta[column-1];

                }else{
                        for (int xi = 0; xi<3; xi++){
                                a_f[xi] = a[column*3+3+xi];
                                a_b[xi] = a[column*3+xi];
                                a_o[xi] = a_dna[ic1+xi];
                        }
                        beta_b = beta_p[ic];
                }

                Bending_force_projection_cuda(g, beta[column], beta_b, length[column], a_f, a_b, a_o, force_projection1, force_projection2, Energy_m[column]);

		force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

		force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda(force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
                torque_due_to_force_cuda(force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

		torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

		torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

	__threadfence();

	//Twisting

	if (column < n-1){

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                for (int xi = 0; xi <3; xi++){
                        r_tmp1[xi] = r[column*3+3+xi];
                        r_tmp2[xi] = r[column*3+xi];
                        a_tmp1[xi] = a[column*3+3+xi];
                        a_tmp2[xi] = a[column*3+xi];
                        b_tmp1[xi] = b[column*3+3+xi];
                        b_tmp2[xi] = b[column*3+xi];
                        c_tmp1[xi] = c[column*3+3+xi];
                        c_tmp2[xi] = c[column*3+xi];
                }

                first_coord_cuda(type[column+1], r_tmp1, a_tmp1, b_tmp1, c_tmp1, r_f);
                second_coord_cuda(type[column], r_tmp2, a_tmp2, b_tmp2, c_tmp2, r_s);

                if (type[column]==0){
                        alpha_b = alpha[column-1];
                        beta_b = beta[column-1];
                        gamma_b = gamma[column-1];
                        phi_o_b = phi_o[column-1];
                        gamma_n = gamma[column-1];
                        for (int xi = 0; xi <3; xi++){
                                b_o[xi] = b[column*3+xi];
                                c_o[xi] = c[column*3+xi];
                        }
                }else{
                        alpha_b = alpha[column];
                        beta_b = beta_p[ic];
                        gamma_b = gamma[column];
                        phi_o_b = phi_o[column];
                        gamma_n = gamma_p[ic];
                        for (int xi = 0; xi <3; xi++){
                                b_o[xi] = b_dna[ic1+xi];
                                c_o[xi] = c_dna[ic1+xi];
                        }
                }

                Twisting_force_projection_cuda(s, alpha[column], beta[column], gamma[column], phi_o[column],  length[column], alpha_b, beta_b, gamma_b, phi_o_b, gamma_n, b_o, c_o, force_projection1, force_projection2, Energy_m[column]);

		force_m[column*2*maxrtlh*3] += force_projection1[0];
                force_m[column*2*maxrtlh*3+1] += force_projection1[1];
                force_m[column*2*maxrtlh*3+2] += force_projection1[2];

		force_m[(column+1)*2*maxrtlh*3+3] += force_projection2[0];
                force_m[(column+1)*2*maxrtlh*3+4] += force_projection2[1];
                force_m[(column+1)*2*maxrtlh*3+5] += force_projection2[2];

                torque_due_to_force_cuda(force_projection1, r_s, r_tmp2, a_tmp2, b_tmp2, c_tmp2, torque1);
                torque_due_to_force_cuda(force_projection2, r_f, r_tmp1, a_tmp1, b_tmp1, c_tmp1, torque2);

		torque_m[column*2*maxrtlh*3] += torque1[0];
                torque_m[column*2*maxrtlh*3+1] += torque1[1];
                torque_m[column*2*maxrtlh*3+2] += torque1[2];

		torque_m[(column+1)*2*maxrtlh*3+3] += torque2[0];
                torque_m[(column+1)*2*maxrtlh*3+4] += torque2[1];
                torque_m[(column+1)*2*maxrtlh*3+5] += torque2[2];

        }

	__threadfence();

	//Mechanical Torques

        if (column < n-1){

                im1 = column-1;
                i1 = column*3;
                i2 = i1+1;
                i3 = i2+1;
                ib1 = i1-3;
                ib2 = ib1+1;
                ib3 = ib2+1;

                ic = 0;
                for (count = 0; count <= column; count++){
                        if(type[count]==1) ic=ic+1;
                }
                ic = ic-1;
                ic1 = 3*ic;
                ic2 = ic1+1;
                ic3 = ic2+1;

                if (type[column]==0){
                        torque[i1] = s*(alpha[column]+gamma[column]+phi_o[column]-alpha[im1]-gamma[im1]-phi_o[im1]);
                        torque[i2] = 0.0;
                        torque[i3] = 0.0;
                }else{
                        ada = a_dna[ic1]*a[i1] + a_dna[ic2]*a[i2] + a_dna[ic3]*a[i3];
                        adb = a_dna[ic1]*b[i1] + a_dna[ic2]*b[i2] + a_dna[ic3]*b[i3];
                        adc = a_dna[ic1]*c[i1] + a_dna[ic2]*c[i2] + a_dna[ic3]*c[i3];
                        
                        mag = s*(alpha[column]+gamma[column]-phi_o[column]);
                        torque[i1] = torque[i1] + mag*ada;
                        torque[i2] = torque[i2] + mag*adb;
                        torque[i3] = torque[i3] + mag*adc;
                        if (column > 0){
                                mag = -s*(alpha[im1]+gamma[im1]-phi_o[im1]);
                                torque[i1] = torque[i1] + mag*co;
                                torque[i2] = torque[i2] + mag*si;
                                torque[i3] = torque[i3] + 0.0;
                        }

                        //Extra Bending torque

                        torque[i2] = torque[i2] - g*beta_p[ic]*adc/sin(beta_p[ic]);
                        torque[i3] = torque[i3] + g*beta_p[ic]*adb/sin(beta_p[ic]);
                
                        if (column > 0){
                                ada = a[ib1]*a[i1] + a[ib2]*a[i2] + a[ib3]*a[i3];
                                adb = a[ib1]*b[i1] + a[ib2]*b[i2] + a[ib3]*b[i3];
                                adc = a[ib1]*c[i1] + a[ib2]*c[i2] + a[ib3]*c[i3];
                                
                                torque[i1] = torque[i1] + g*beta[im1]*(si*adc)/sin(beta[im1]);
                                torque[i2] = torque[i2] - g*beta[im1]*(co*adc)/sin(beta[im1]);
                                torque[i3] = torque[i3] + g*beta[im1]*(co*adb-si*ada)/sin(beta[im1]);
                        }

                        //Extra Twisting torque

                        s1 = sin(alpha_p[ic]);
                        c1 = cos(alpha_p[ic]);
                        mag = s*(alpha[column]+gamma[column]-phi_o[column])*tan(0.5*beta_p[ic]);
                        cda = c_dna[ic1]*a[i1] + c_dna[ic2]*a[i2] + c_dna[ic3]*a[i3];
                        bda = b_dna[ic1]*a[i1] + b_dna[ic2]*a[i2] + b_dna[ic3]*a[i3];
                        cdb = c_dna[ic1]*b[i1] + c_dna[ic2]*b[i2] + c_dna[ic3]*b[i3];
                        bdb = b_dna[ic1]*b[i1] + b_dna[ic2]*b[i2] + b_dna[ic3]*b[i3];
                        cdc = c_dna[ic1]*c[i1] + c_dna[ic2]*c[i2] + c_dna[ic3]*c[i3];
                        bdc = b_dna[ic1]*c[i1] + b_dna[ic2]*c[i2] + b_dna[ic3]*c[i3];

                        torque[i1] = torque[i1] - mag*(s1*cda + c1*bda);
                        torque[i2] = torque[i2] - mag*(s1*cdb + c1*bdb);
                        torque[i3] = torque[i3] - mag*(s1*cdc + c1*bdc);

                        if (column > 0){
                                s1 = sin(gamma[im1]);
                                c1 = cos(gamma[im1]);
                                mag = s*(alpha[im1]+gamma[im1]-phi_o[im1])*tan(0.5*beta[im1]);
                                cda = 0.0;
                                bda = -si;
                                cdb = 0.0;
                                bdb = co;
                                cdc = 1.0;
                                bdc = 0.0;
                                
                                torque[i1] = torque[i1] - mag*(s1*cda - c1*bda);
                                torque[i2] = torque[i2] - mag*(s1*cdb - c1*bdb);
                                torque[i3] = torque[i3] - mag*(s1*cdc - c1*bdc);
                        }

                }
        
        }

	__threadfence();

	//Additional torque for last bead

        if (column==0){

                torque[n3-3] = -s*(alpha[n-2]+gamma[n-2]-phi_o[n-2]);
                torque[n3-2] = 0.0;
                torque[n3-1] = 0.0;

        }


}

__global__ void init_force_torque_m(int maxrtlh, double* force_m, double* torque_m, double* t_force_m, double* LH_force_m, double* temp_force_m, double* Energy_m){

	// Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < maxrtlh*2 && j < maxrtlh*3){
                force_m[i*maxrtlh*3+j] = 0.0;
                torque_m[i*maxrtlh*3+j] = 0.0;
		t_force_m[i*maxrtlh*3+j] = 0.0;
		LH_force_m[i*maxrtlh*3+j] = 0.0;
		temp_force_m[i*maxrtlh*3+j] = 0.0;
        }

	if (i < maxrtlh && j< maxrtlh){
                Energy_m[i*maxrtlh+j]=0.0;
        }

}


__global__ void Electrostatic_and_Excluded_volume_force_cuda(int maxrtlh, int n, int n3, int n_c, int nc3, int* type, double* r, double* a, double* b, double* c, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, int Nq, int Nq3, double* core_pos, double* core_q, double* force, double* torque, double* Energy, double* force_m, double* torque_m, double* Energy_m){

	double ql_ql, dist;
        int k, l, ch;
        int i1, i2, i3, j1, j2, j3, k1, k2, k3, l1, l2, l3;
        double mi, Rcut, temp, temp1, temp2;
        double z[3];
        double fa, fb, fc;
        double g1, s1, s2;
	

	Rcut = 25.0;
        ql_ql = q_l*q_l;

	// Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n-1){
                i1 = 3*i;
                i2 = i1+1;
                i3 = i2+1;

		if (j >=  i+1 and j < n){
                        j1 = j*3;
                        j2 = j1+1;
                        j3 = j2+1;

                        dist = (r[j1]-r[i1])*(r[j1]-r[i1])+(r[j2]-r[i2])*(r[j2]-r[i2])+(r[j3]-r[i3])*(r[j3]-r[i3]);
			dist = sqrt(dist);



                        ch = 1;
                        if (dist > Rcut) ch=0;
                        if (type[i+1] != 0) ch=0;
                        if (i > 0 and type[i-1] != 0) ch=0;
                        if (j < n-1){
                                if (type[j+1] != 0) ch=0;
                        }
                        if (type[j-1] != 0) ch=0;


			if (ch == 1){
                                if (type[i] == 0){
                                        if (type[j] == 0){
						if (abs(i-j) > 1){
							mi = 1.0/dist;
                                                        z[0] = mi*(r[i1]-r[j1]);
                                                        z[1] = mi*(r[i2]-r[j2]);
                                                        z[2] = mi*(r[i3]-r[j3]);

                                                        temp = -debyell*dist;

                                                        g1 = k_e*ql_ql*exp(temp)*(debyell*dist+1.0)/(dist*dist);

							Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*ql_ql/dist)*exp(temp);
							if (dist <= 8){
                                                                s1 = sigma_DNA_DNA;
                                                                s2 = sigma_DNA_DNA;
                                                                temp1 = s1/dist;
                                                                temp2 = s2/dist;
                                                                g1 = g1 + k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));
                                                        	Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));
                                                        }

							force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
							force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
							force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
							force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
							force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
							force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

						}
					}else{
						//DNA-Core interaction

                                                for (k=0;k<Nq;k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        z[0] = (r[i1]-(r[j1]+a[j1]*core_pos[k1]+b[j1]*core_pos[k2]+c[j1]*core_pos[k3]));
                                                        z[1] = (r[i2]-(r[j2]+a[j2]*core_pos[k1]+b[j2]*core_pos[k2]+c[j2]*core_pos[k3]));
                                                        z[2] = (r[i3]-(r[j3]+a[j3]*core_pos[k1]+b[j3]*core_pos[k2]+c[j3]*core_pos[k3]));
                                                        dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
							dist = sqrt(dist);
                                                        mi = 1.0/dist;
                                                        z[0] = mi*z[0];
                                                        z[1] = mi*z[1];
                                                        z[2] = mi*z[2];

                                                        if (abs(i-j)>1){

								temp = -debyell*dist;
                                                                g1 = k_e*q_l*core_q[k]*exp(temp)*(debye*dist+1.0)/(dist*dist);
                                                        	Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*q_l*core_q[k]/dist)*exp(temp);

								force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                	                        force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                        	                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                	                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
        	                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
	                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

								//torque due to force
                                                                fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
								fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

								torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[k1] - fa*core_pos[k2];



							}

							// Excluded Volume force

	                                                if (dist <= 8.0 and core_q[k]>0){
        	                                                s1 = sigma_DNA_Core;
                	                                        s2 = sigma_DNA_Core;
								temp1 = s1/dist;
	                                                        temp2 = s2/dist;
                       	                                        g1 = k_ex*((12.0/s1)*pow((temp1),13)-(6.0/s2)*pow((temp2),7));
								Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));


								force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
								force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

	                                                        //torque due to force
        	                                                fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
               	                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                       	                                        fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

								torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

                                                        }
							
						}
					}
				
				}else{

                                        //Core-DNA interaction
                                        if (type[j] == 0){
                                                for (k=0;k<Nq;k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        z[0] = (-r[j1]+(r[i1]+a[i1]*core_pos[k1]+b[i1]*core_pos[k2]+c[i1]*core_pos[k3]));
                                                        z[1] = (-r[j2]+(r[i2]+a[i2]*core_pos[k1]+b[i2]*core_pos[k2]+c[i2]*core_pos[k3]));
                                                        z[2] = (-r[j3]+(r[i3]+a[i3]*core_pos[k1]+b[i3]*core_pos[k2]+c[i3]*core_pos[k3]));
                                                        dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
                                                        dist = sqrt(dist);

                                                        mi = 1.0/dist;
                                                        z[0] = mi*z[0];
                                                        z[1] = mi*z[1];
                                                        z[2] = mi*z[2];
                                                        if (abs(i-j) > 1){
                                                                temp = -debye*dist;
                                                                g1 = k_e*q_l*core_q[k]*exp(temp)*(debye*dist+1.0)/(dist*dist);
								Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*q_l*core_q[k]/dist)*exp(temp);

								force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];

                                                                //torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);
								
								torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

							}

                                                        //Excluded Volume force
                                                        if (dist <= 8.0 and core_q[k]>0){
                                                                s1 = sigma_DNA_Core;
                                                                s2 = sigma_DNA_Core;
                                                                temp1 = s1/dist;
                                                                temp2 = s2/dist;

								Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));
                                                                g1 = k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));

								force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];
                                                                
								//torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

								torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

							}
                                                }
					}else{//Core-Core interaction

                                                for (k = 0; k < Nq; k++){
                                                        k1 = 3*k;
                                                        k2 = k1+1;
                                                        k3 = k2+1;
                                                        for (l = 0; l < Nq; l++){
                                                                l1 = 3*l;
                                                                l2 = l1+1;
                                                                l3 = l2+1;
                                                                z[0] = (r[i1]+a[i1]*core_pos[k1]+b[i1]*core_pos[k2]+c[i1]*core_pos[k3] - (r[j1]+a[j1]*core_pos[l1]+b[j1]*core_pos[l2]+c[j1]*core_pos[l3]));
                                                                z[1] = (r[i2]+a[i2]*core_pos[k1]+b[i2]*core_pos[k2]+c[i2]*core_pos[k3] - (r[j2]+a[j2]*core_pos[l1]+b[j2]*core_pos[l2]+c[j2]*core_pos[l3]));
                                                                z[2] = (r[i3]+a[i3]*core_pos[k1]+b[i3]*core_pos[k2]+c[i3]*core_pos[k3] - (r[j3]+a[j3]*core_pos[l1]+b[j3]*core_pos[l2]+c[j3]*core_pos[l3]));
                                                                dist = z[0]*z[0]+z[1]*z[1]+z[2]*z[2];
                                                                dist = sqrt(dist);

                                                                mi = 1.0/dist;
                                                                z[0] = mi*z[0];
                                                                z[1] = mi*z[1];
                                                                z[2] = mi*z[2];

                                                                temp = -debye*dist;
                                                                g1 = k_e*core_q[k]*core_q[l]*exp(temp)*(debye*dist+1.0)/(dist*dist);
								Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + (k_e*core_q[l]*core_q[k]/dist)*exp(temp);

								force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];
  
                                                                //torque due to force
                                                                fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
								fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

								torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

								fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);

								torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[l2] - fb*core_pos[l3];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[l3] - fc*core_pos[l1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[l1] - fa*core_pos[l2];

                                                                //Excluded Volume force
                                                                if (dist <= 8.0){
                                                                        s1 = sigma_Core_Core;
                                                                        s2 = sigma_Core_Core;
                                                                        temp1 = s1/dist;
                                                                        temp2 = s2/dist;
                                                                        g1 = k_ex*((12.0/s1)*pow(temp1,13)-(6.0/s2)*pow(temp2,7));

									Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + k_ex*(pow(temp1,12) - pow(temp2,6));

									force_m[i*2*maxrtlh*3+j*3] += g1*z[0];
                                                                	force_m[i*2*maxrtlh*3+j*3+1] += g1*z[1];
	                                                                force_m[i*2*maxrtlh*3+j*3+2] += g1*z[2];
        	                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] -= g1*z[0];
                	                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] -= g1*z[1];
                        	                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] -= g1*z[2];


                                                                        //torque due to force
                                                                        fa = g1*(a[i1]*z[0]+a[i2]*z[1]+a[i3]*z[2]);
                                                                        fb = g1*(b[i1]*z[0]+b[i2]*z[1]+b[i3]*z[2]);
                                                                        fc = g1*(c[i1]*z[0]+c[i2]*z[1]+c[i3]*z[2]);

									torque_m[i*2*maxrtlh*3+j*3] = torque_m[i*2*maxrtlh*3+j*3] + fc*core_pos[k2] - fb*core_pos[k3];
        	                                                        torque_m[i*2*maxrtlh*3+j*3+1] = torque_m[i*2*maxrtlh*3+j*3+1] + fa*core_pos[k3] - fc*core_pos[k1];
	                                                                torque_m[i*2*maxrtlh*3+j*3+2] = torque_m[i*2*maxrtlh*3+j*3+2] + fb*core_pos[k1] - fa*core_pos[k2];

									fa = -g1*(a[j1]*z[0]+a[j2]*z[1]+a[j3]*z[2]);
                                                                        fb = -g1*(b[j1]*z[0]+b[j2]*z[1]+b[j3]*z[2]);
                                                                        fc = -g1*(c[j1]*z[0]+c[j2]*z[1]+c[j3]*z[2]);
									
									torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] + fc*core_pos[l2] - fb*core_pos[l3];
        	                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] + fa*core_pos[l3] - fc*core_pos[l1];
  	     	                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] = torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] + fb*core_pos[l1] - fa*core_pos[l2];

								}

                                                        }
                                                }

                                        }
				}
					
			}


                        }



                


        }
}

__global__ void step_init(int n, int n_tail, int n_LH, double* force, double* force_n, double* torque, double* torque_n, double* t_force, double* t_force_n, double* LH_force, double* LH_force_n, double* Energy, double* r_all, double* r, double* r_t, double* r_lh){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	Energy[0] = 0.0;
	if (i < n*3){
		force[i]=0;
                force_n[i]=0;
                torque[i]=0;
                torque_n[i]=0;
	}
	if (i < n_tail*3){
		t_force[i] = 0;
		t_force_n[i] = 0;
	}
	if (i < n_LH*3){
		LH_force[i] = 0;
		LH_force_n[i] = 0;
	}

	if (i<n){
		r_all[i*3] = r[i*3];
		r_all[i*3+1] = r[i*3+1];
		r_all[i*3+2] = r[i*3+2];
	}else if (i<n+n_tail){
		r_all[i*3] = r_t[i*3-n*3];
                r_all[i*3+1] = r_t[i*3-n*3+1];
                r_all[i*3+2] = r_t[i*3-n*3+2];
	}else if (i<n+n_tail+n_LH){
		r_all[i*3] = r_lh[i*3-n*3-n_tail*3];
                r_all[i*3+1] = r_lh[i*3-n*3-n_tail*3+1];
                r_all[i*3+2] = r_lh[i*3-n*3-n_tail*3+2];
	}

}

__global__ void force_reduction(int n, int n_tail, int n_LH, int maxrtlh, int maxrtlh3, double* force_m, double* force, double* torque_m, double* torque, double* t_force_m, double* t_force, double* LH_force_m, double* LH_force){

        int column = blockIdx.x * blockDim.x + threadIdx.x;
        int index1, index2;

        if (column < maxrtlh){
                for (int i=0; i<maxrtlh; i++){
                        index1 = column*2*maxrtlh3+i*3;
                        index2 = column*2*maxrtlh3+maxrtlh3+i*3;
			if (column<n){
				force[column*3] += force_m[index1] + force_m[index2];
                        	force[column*3+1] += force_m[index1+1] + force_m[index2+1];
                        	force[column*3+2] += force_m[index1+2] + force_m[index2+2];
			}
			if (column<n){
				torque[column*3] += torque_m[index1] + torque_m[index2];
                        	torque[column*3+1] += torque_m[index1+1] + torque_m[index2+1];
                        	torque[column*3+2] += torque_m[index1+2] + torque_m[index2+2];
			}
			if (column<n_tail){
				t_force[column*3] += t_force_m[index1] + t_force_m[index2];
                        	t_force[column*3+1] += t_force_m[index1+1] + t_force_m[index2+1];
                        	t_force[column*3+2] += t_force_m[index1+2] + t_force_m[index2+2];
			}
			if (column< n_LH){
				LH_force[column*3] += LH_force_m[index1] + LH_force_m[index2];
                       		LH_force[column*3+1] += LH_force_m[index1+1] + LH_force_m[index2+1];
                        	LH_force[column*3+2] += LH_force_m[index1+2] + LH_force_m[index2+2];
			}
                }
        }

}

__global__ void Energy_reduction(int maxrtlh, double* Energy, double* Energy_m){
        int column = blockIdx.x * blockDim.x + threadIdx.x;

        if (column< maxrtlh){
                for (int i=1; i<maxrtlh; i++){
                        Energy_m[column] += Energy_m[i*maxrtlh+column];
                }
        }

        __threadfence();

        if (column==0){
                for (int i=0; i<maxrtlh; i++){
                        Energy[0] += Energy_m[i];
                }
        }


}

__global__ void update_tail_beta_cuda(int n_tail, double* r_t, double* beta_t, int* t_grp){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        double x12, y12, z12, r12, x32, y32, z32, r32, p123;

        if (i < n_tail){
                if((t_grp[i] == t_grp[i+1]) and (t_grp[i] == t_grp[i+2]) and (i < n_tail-2)){
                        x12 = r_t[i*3] - r_t[i*3+3];
                        y12 = r_t[i*3+1] - r_t[i*3+4];
                        z12 = r_t[i*3+2] - r_t[i*3+5];

                        r12 = x12*x12 + y12*y12 + z12*z12;
			r12 = sqrt(r12);

                        x32 = r_t[i*3+6] - r_t[i*3+3];
                        y32 = r_t[i*3+7] - r_t[i*3+4];
                        z32 = r_t[i*3+8] - r_t[i*3+5];

                        r32 = x32*x32 + y32*y32 + z32*z32;
			r32 = sqrt(r32);

                        p123 = (x12*x32 + y12*y32 + z12*z32)/(r12*r32);

                        beta_t[i] = acos(p123);
                }else{
                        beta_t[i] = 0.0;
                }

        }

}


__global__ void tail_force_cuda_str_ben(int maxrtlh, int n, int n_t, int* type, int n_tail, int n_tail3, double* tail_pos, int* tail_fix, int* nc_t_flag, double* r_t, double* beta_t, double* h_t, double* g_t, double* lo_t, double* beta_o_t, double h, double k_e, double debye, double k_ex, double* t_q, double* t_rad, int* t_grp, int* t_fix, double* r, double* a, double* b, double* c, double q_l, int Nq, int Nq3, double* core_pos, double* core_q, double* t_force, double* force, double* torque, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, double* Energy, double* t_force_m, double* Energy_m){

        double force_projection1[3], force_projection2[3],force_projection3[3];
        double r_t_tmp1[3], r_t_tmp2[3], r_t_tmp3[3];

	// Calculate Row and Column
        int i = blockIdx.x * blockDim.x + threadIdx.x;

	//tail stretching
	if (i<n_tail-1){
                if (t_grp[i]==t_grp[i+1]){
                        for (int ix = 0; ix<3; ix++){
                                r_t_tmp1[ix] = r_t[i*3+ix];
                                r_t_tmp2[ix] = r_t[i*3+3+ix];
                        }
                        Force_Stretching_cuda(h_t[i], r_t_tmp1, r_t_tmp2, lo_t[i], force_projection1, force_projection2, Energy_m[i]);

                        t_force_m[i*2*maxrtlh*3] += force_projection1[0];
                        t_force_m[i*2*maxrtlh*3+1] += force_projection1[1];
                        t_force_m[i*2*maxrtlh*3+2] += force_projection1[2];

			t_force_m[(i+1)*2*maxrtlh*3+3] += force_projection2[0];
                        t_force_m[(i+1)*2*maxrtlh*3+4] += force_projection2[1];
                        t_force_m[(i+1)*2*maxrtlh*3+5] += force_projection2[2];

                }
        }

	__threadfence();
	
	//tail bending
	if (i<n_tail-2){
                if ((t_grp[i]==t_grp[i+1]) and (t_grp[i]==t_grp[i+2])){
                        for (int ix = 0; ix<3; ix++){
                                r_t_tmp1[ix] = r_t[i*3+ix];
                                r_t_tmp2[ix] = r_t[i*3+3+ix];
                                r_t_tmp3[ix] = r_t[i*3+6+ix];
                        }
                        Force_Bending_cuda(g_t[i],beta_t[i],beta_o_t[i], r_t_tmp1, r_t_tmp2, r_t_tmp3, force_projection1, force_projection2, force_projection3, Energy_m[i]);

			t_force_m[i*2*maxrtlh*3] -= force_projection1[0];
                        t_force_m[i*2*maxrtlh*3+1] -= force_projection1[1];
                        t_force_m[i*2*maxrtlh*3+2] -= force_projection1[2];

			t_force_m[(i+1)*2*maxrtlh*3+3] -= force_projection2[0];
                        t_force_m[(i+1)*2*maxrtlh*3+4] -= force_projection2[1];
                        t_force_m[(i+1)*2*maxrtlh*3+5] -= force_projection2[2];

			t_force_m[(i+2)*2*maxrtlh*3+6] -= force_projection3[0];
                        t_force_m[(i+2)*2*maxrtlh*3+7] -= force_projection3[1];
                        t_force_m[(i+2)*2*maxrtlh*3+8] -= force_projection3[2];

                }
        }

	__threadfence();

}

__global__ void tail_force_cuda_ele_vdw(int maxrtlh, int n, int n_t, int* type, int n_tail, int n_tail3, double* tail_pos, int* tail_fix, int* nc_t_flag, double* r_t, double* beta_t, double* h_t, double* g_t, double* lo_t, double* beta_o_t, double h, double k_e, double debye, double k_ex, double* t_q, double* t_rad, int* t_grp, int* t_fix, double* r, double* a, double* b, double* c, double q_l, int Nq, int Nq3, double* core_pos, double* core_q, double* t_force, double* force, double* torque, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, double* Energy, double* t_force_m, double* force_m, double* torque_m, double* Energy_m){

	double Stri[3];
        int k;
        double r_to[3];
        double force_projection1[3], force_projection2[3];
        double torque_tc[3];

        double r_t_tmp1[3], r_t_tmp2[3], r_tmp[3], tail_pos_tmp[3], a_tmp[3], b_tmp[3], c_tmp[3], h_stri_tmp[3];

        // Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i<n_tail-1){
                if (j>i and j<n_tail){
			
                        if ((t_grp[i]!=t_grp[j]) or (j>i+2)){
                                for (int ix = 0; ix<3; ix++){
                                        r_t_tmp1[ix] = r_t[i*3+ix];
                                        r_t_tmp2[ix] = r_t[j*3+ix];
                                }
                                Force_Electrostatics_cuda(t_q[i], t_q[j], 1/(4*PI*k_e), debye, r_t_tmp1, r_t_tmp2, force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);
				
				t_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                t_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                t_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

				
				t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

			}

                }
        }

	__threadfence();


	if (i<n_tail){
                if (j<n){
			if(type[j]==0){
				
                                for (int ix = 0; ix<3; ix++){
                                        r_t_tmp1[ix] = r_t[i*3+ix];
                                        r_tmp[ix] = r[j*3+ix];
                                }

				Force_Ele_Vdw_cuda(t_q[i], q_l, 1/(4*PI*k_e), debye,k_ex, sigma_Tail_Linker, r_t_tmp1, r_tmp, force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

                                t_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                t_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                t_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

			}

		}

	}
	__threadfence();


	//Stretching between tail and core

        if  (i < n){
                if (type[i]==1){
			k=0;
			for (int cnti=0; cnti<i;cnti++){
				if (type[cnti]==1){
					k=k+1;
				}
			}
                        if(nc_t_flag[k]==1){
                                if (j < n_t){
                                        if (tail_fix[j]==1){

                                                r_to[0] = r[i*3] + a[i*3]*tail_pos[j*3] + b[i*3]*tail_pos[j*3+1] + c[i*3]*tail_pos[j*3+2];
                                                r_to[1] = r[i*3+1] + a[i*3+1]*tail_pos[j*3] + b[i*3+1]*tail_pos[j*3+1] + c[i*3+1]*tail_pos[j*3+2];
                                                r_to[2] = r[i*3+2] + a[i*3+2]*tail_pos[j*3] + b[i*3+2]*tail_pos[j*3+1] + c[i*3+2]*tail_pos[j*3+2];

                                                Stri[0] = r_t[k*n_t*3+j*3] - r_to[0];
                                                Stri[1] = r_t[k*n_t*3+j*3+1] - r_to[1];
                                                Stri[2] = r_t[k*n_t*3+j*3+2] - r_to[2];

                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + h*(Stri[0]*Stri[0]+Stri[1]*Stri[1]+Stri[2]*Stri[2])/2;

                                                t_force_m[(k*n_t+j)*2*maxrtlh*3+i*3] -= h*Stri[0];
                                                t_force_m[(k*n_t+j)*2*maxrtlh*3+i*3+1] -= h*Stri[1];
                                                t_force_m[(k*n_t+j)*2*maxrtlh*3+i*3+2] -= h*Stri[2];

                                                force_m[i*2*maxrtlh*3+j*3] += h*Stri[0];
                                                force_m[i*2*maxrtlh*3+j*3+1] += h*Stri[1];
                                                force_m[i*2*maxrtlh*3+j*3+2] += h*Stri[2];

                                                // torque due to force

                                                for (int ix = 0; ix<3; ix++){
                                                        tail_pos_tmp[ix] =  tail_pos[j*3+ix];
                                                        a_tmp[ix] = a[i*3+ix];
                                                        b_tmp[ix] = b[i*3+ix];
                                                        c_tmp[ix] = c[i*3+ix];
                                                        h_stri_tmp[ix] = h*Stri[ix];
                                                }

                                                torque_due_to_force_relative_cuda(h_stri_tmp, tail_pos_tmp, a_tmp, b_tmp, c_tmp, torque_tc);
                                                torque_m[i*2*maxrtlh*3+j*3] += torque_tc[0];
                                                torque_m[i*2*maxrtlh*3+j*3+1] += torque_tc[1];
                                                torque_m[i*2*maxrtlh*3+j*3+2] += torque_tc[2];

                                        }
                                }
                        }
                }
        }

}

__global__ void update_LH_beta_cuda(int n_LH, int n_lh_c, double* r_lh, double* beta_lh){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        double x12, y12, z12, r12, x32, y32, z32, r32, p123;

        if (i < n_LH){
                if((i%n_lh_c!=0) and (i%n_lh_c != n_lh_c-1)){
                        x12 = r_lh[i*3-3] - r_lh[i*3];
                        y12 = r_lh[i*3-2] - r_lh[i*3+1];
                        z12 = r_lh[i*3-1] - r_lh[i*3+2];

                        r12 = x12*x12 + y12*y12 + z12*z12;
			r12 = sqrt(r12);

                        x32 = r_lh[i*3+3] - r_lh[i*3];
                        y32 = r_lh[i*3+4] - r_lh[i*3+1];
                        z32 = r_lh[i*3+5] - r_lh[i*3+2];

                        r32 = x32*x32 + y32*y32 + z32*z32;
			r32 = sqrt(r32);

                        p123 = (x12*x32 + y12*y32 + z12*z32)/(r12*r32);

                        beta_lh[i] = acos(p123);
                }else{
                        beta_lh[i] = 0.0;
                }

        }

}

__global__ void Linker_Histone_str_ben_cuda(int maxrtlh, int n, int n_tail, int n_lh_n, int n_lh_g, int n_lh_c, int* t_grp, int* t_fix, int* type, int n_LH, int n_LH3, double* LH_g_pos, int* LH_conn, int* nc_lh_flag, double* beta_lh, double* r_lh, double* LH_q, double k_e, double debye, double k_ex, double* LH_vdw_hh, double* LH_vdw_hc, double* LH_vdw_hl, double* LH_vdw_ht, double* LH_kstr, double* LH_kben, double* LH_streq, double* LH_betaeq, double* r, double* a, double* b, double* c, double q_l, int Nq, int Nq3, double* core_pos, double* core_q, double* r_t, double* t_q, double* t_force, double* LH_force, double* force, double* torque, double* Energy, double* LH_force_m, double* Energy_m){

        int xi;
        int index;
        double force_projection1[3], force_projection2[3],force_projection3[3];

        double r_lh_tmp1[3], r_lh_tmp2[3], r_lh_tmp3[3];

	// Calculate Row and Column
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        //Stretching force between LH beads

        if(i<n_LH){
                index = i%(n_lh_n+n_lh_c);
                if (index < n_lh_n){
                        index = index;
                }else{
                        index = index + n_lh_g;
                }

                if (LH_conn[index]==1){
                        for (xi = 0; xi<3; xi++){
                                r_lh_tmp1[xi] = r_lh[i*3+xi];
                                r_lh_tmp2[xi] = r_lh[i*3+3+xi];
                        }

                        Force_Stretching_cuda(LH_kstr[index], r_lh_tmp1, r_lh_tmp2, LH_streq[index],force_projection1, force_projection2, Energy_m[i]);
                        LH_force_m[i*2*maxrtlh*3] += force_projection1[0];
                        LH_force_m[i*2*maxrtlh*3+1] += force_projection1[1];
                        LH_force_m[i*2*maxrtlh*3+2] += force_projection1[2];

			LH_force_m[(i+1)*2*maxrtlh*3+3] += force_projection2[0];
                        LH_force_m[(i+1)*2*maxrtlh*3+4] += force_projection2[1];
                        LH_force_m[(i+1)*2*maxrtlh*3+5] += force_projection2[2];
                }
        }

	__threadfence();
	
	//Bending between LH beads

        if(i<n_LH){
                index = i%(n_lh_n+n_lh_c);
                if (index < n_lh_n){
                        index = index;
                }else{
                        index = index + n_lh_g;
                }
                if ((i%n_lh_c!=0) and (i%n_lh_c!=n_lh_c-1)){
                        for (xi = 0; xi<3; xi++){
                                r_lh_tmp1[xi] = r_lh[i*3-3+xi];
                                r_lh_tmp2[xi] = r_lh[i*3+xi];
                                r_lh_tmp3[xi] = r_lh[i*3+3+xi];
                        }
                        Force_Bending_cuda(LH_kben[index],beta_lh[i],LH_betaeq[index],r_lh_tmp1, r_lh_tmp2, r_lh_tmp3,force_projection1,force_projection2,force_projection3, Energy_m[i]);
			
			LH_force_m[(i-1)*2*maxrtlh*3] -= force_projection1[0];
                        LH_force_m[(i-1)*2*maxrtlh*3+1] -= force_projection1[1];
                        LH_force_m[(i-1)*2*maxrtlh*3+2] -= force_projection1[2];

			LH_force_m[i*2*maxrtlh*3+3] -= force_projection2[0];
                        LH_force_m[i*2*maxrtlh*3+4] -= force_projection2[1];
                        LH_force_m[i*2*maxrtlh*3+5] -= force_projection2[2];

                        LH_force_m[(i+1)*2*maxrtlh*3+6] -= force_projection3[0];
                        LH_force_m[(i+1)*2*maxrtlh*3+7] -= force_projection3[1];
                        LH_force_m[(i+1)*2*maxrtlh*3+8] -= force_projection3[2];	


                }
        }

	__threadfence();
}

__global__ void temp_force_2_force_m(int n, int maxrtlh, int* t_grp, int* type, double* temp_force_m, double* force_m, double* torque_m){

	int k, index_c;

	// Calculate Row and Column
        int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < maxrtlh){
		k=0;
        	for (int ccnt=0; ccnt<n; ccnt++){
	        	if (k==(t_grp[j]-1)/10 and type[ccnt]==1){
                		index_c = ccnt;
                        	break;
                	}
                	if(type[ccnt]==1){
                        	k=k+1;
                	}
        	}
		for (int i=0; i< maxrtlh; i++){
			force_m[index_c*2*maxrtlh*3+j*3] += temp_force_m[i*2*maxrtlh*3+j*3];
			force_m[index_c*2*maxrtlh*3+j*3+1] += temp_force_m[i*2*maxrtlh*3+j*3+1];
			force_m[index_c*2*maxrtlh*3+j*3+2] += temp_force_m[i*2*maxrtlh*3+j*3+2];
			torque_m[index_c*2*maxrtlh*3+j*3] += temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3];
                        torque_m[index_c*2*maxrtlh*3+j*3+1] += temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+1];
                        torque_m[index_c*2*maxrtlh*3+j*3+2] += temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+2];
		}
	}


}

__global__ void Linker_Histone_ele_vdw_cuda(int maxrtlh, int n, int n_tail, int n_lh_n, int n_lh_g, int n_lh_c, int* t_grp, int* t_fix, int* type, int n_LH, int n_LH3, double* LH_g_pos, int* LH_conn, int* nc_lh_flag, double* beta_lh, double* r_lh, double* LH_q, double k_e, double debye, double k_ex, double* LH_vdw_hh, double* LH_vdw_hc, double* LH_vdw_hl, double* LH_vdw_ht, double* LH_kstr, double* LH_kben, double* LH_streq, double* LH_betaeq, double* r, double* a, double* b, double* c, double q_l, int Nq, int Nq3, double* core_pos, double* core_q, double* r_t, double* t_q, double* t_force, double* LH_force, double* force, double* torque, double* Energy, double* LH_force_m, double* force_m, double* torque_m, double* t_force_m, double* temp_force_m, double* Energy_m){
	
	double Stri[3];
        double distance, fa, fb, fc;
        int k, l, k2, m, m1, xi, count;
        double r_lho[3], r_lho2[3], z[3];
        int index, index1, index2, index_c;
        int nlh = n_lh_n+n_lh_c;
        double torque_hgl[3], torque1[3], torque2[3];
        double force_projection1[3], force_projection2[3];

        double r_lh_tmp1[3], r_lh_tmp2[3], r_t_tmp[3], r_tmp[3], a_tmp[3], b_tmp[3], c_tmp[3], core_pos_tmp[3], LH_g_pos_tmp[3];

	//Electrostatic and Excluded Volume between LH

	// Calculate Row and Column
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        
	if(i<n_LH){
                index1 = i%(n_lh_n+n_lh_c);
                if (index1 < n_lh_n){
                        index1 = index1;
                }else{
                        index1 = index1 + n_lh_g;
                }
                if (j >=i and j<n_LH){
                        index2 = j%(n_lh_n+n_lh_c);
                        if (index2 < n_lh_n){
                                index2 = index2;
                        }else{
                                index2 = index2 + n_lh_g;
                        }

                        if ((i/(n_lh_n+n_lh_c) == j/(n_lh_n+n_lh_c)) and (j > i+1)){

                                for (xi = 0; xi<3; xi++){
                                        r_lh_tmp1[xi] = r_lh[i*3+xi];
                                        r_lh_tmp2[xi] = r_lh[j*3+xi];
                                }

                                Force_Exclude_Volume_cuda(k_ex, (LH_vdw_hh[index1]+LH_vdw_hh[index2])/2, r_lh_tmp1, r_lh_tmp2,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

				LH_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                LH_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                LH_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

				LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                        }else if ((i/(n_lh_n+n_lh_c) != j/(n_lh_n+n_lh_c))){
                                for (xi = 0; xi<3; xi++){
                                        r_lh_tmp1[xi] = r_lh[i*3+xi];
                                        r_lh_tmp2[xi] = r_lh[j*3+xi];
                                }


                                Force_Ele_Vdw_cuda(LH_q[index1], LH_q[index2], 1/(4*PI*k_e), debye,k_ex, (LH_vdw_hh[index1]+LH_vdw_hh[index2])/2, r_lh_tmp1, r_lh_tmp2,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

				LH_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                LH_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                LH_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];
				
				LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                        }

                }
        }

	__threadfence();

	//Electrostatic and Excluded Volume between LH and DNA

        if (i<n_LH){
                index = i%(n_lh_n+n_lh_c);
                if (index < n_lh_n){
                        index = index;
                }else{
                        index = index + n_lh_g;
                }
                if (j<n){
                        if (type[j]==0){
                                for (xi = 0; xi<3; xi++){
                                        r_lh_tmp1[xi] = r_lh[i*3+xi];
                                        r_tmp[xi] = r[j*3+xi];
                                }
                                Force_Ele_Vdw_cuda(LH_q[index], q_l, 1/(4*PI*k_e), debye,k_ex, LH_vdw_hl[index], r_lh_tmp1, r_tmp,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);
				LH_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                LH_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                LH_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

				force_m[j*2*maxrtlh*3+i*3] += force_projection2[0];
                                force_m[j*2*maxrtlh*3+i*3+1] += force_projection2[1];
                                force_m[j*2*maxrtlh*3+i*3+2] += force_projection2[2];

                        }
                }
        }

	__threadfence();

	//Electrostatic and Excluded Volume between LH and Tails

        if (i<n_LH){
                index = i%(n_lh_n+n_lh_c);
                if (index < n_lh_n){
                        index = index;
                }else{
                        index = index + n_lh_g;
                }
                if (j<n_tail){
                        for (xi = 0; xi<3; xi++){
                                r_lh_tmp1[xi] = r_lh[i*3+xi];
                                r_t_tmp[xi] = r_t[j*3+xi];
                        }
                        Force_Ele_Vdw_cuda(LH_q[index], t_q[j], 1/(4*PI*k_e), debye,k_ex, LH_vdw_ht[index], r_lh_tmp1, r_t_tmp,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);
                        LH_force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                        LH_force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                        LH_force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

                        if (t_fix[j]==0){
                                t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];
                        }else{
                                k=0;
                                for (int ccnt=0; ccnt<n; ccnt++){
                                        if (k==(t_grp[j]-1)/10 and type[ccnt]==1){
                                                index_c = ccnt;
                                                break;
                                        }
                                        if(type[ccnt]==1){
                                                k=k+1;
                                        }
                                }

                                for (int ix = 0; ix<3; ix++){
                                        r_tmp[ix] = r[index_c*3+ix];
                                        a_tmp[ix] = a[index_c*3+ix];
                                        b_tmp[ix] = b[index_c*3+ix];
                                        c_tmp[ix] = c[index_c*3+ix];
                                }
                                torque_due_to_force_cuda(force_projection2, r_t_tmp, r_tmp, a_tmp, b_tmp, c_tmp, torque2);
				
				temp_force_m[i*2*maxrtlh*3+j*3] += force_projection2[0];
				temp_force_m[i*2*maxrtlh*3+j*3+1] += force_projection2[1];
				temp_force_m[i*2*maxrtlh*3+j*3+2] += force_projection2[2];
				temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3] += torque2[0];
                                temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+1] += torque2[1];
                                temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+2] += torque2[2];
                        }

                }
        }

	__threadfence();

	//Electrostatic and Excluded Volume between LH global head and Tails

        if (i<n){
                if (type[i]==1){
			k2=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k2=k2+1;
                                }
                        }
                        if (nc_lh_flag[k2]==1){
                                for (m=0;m<n_lh_g;m++){
                                        r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[m*3] + b[i*3]*LH_g_pos[m*3+1] + c[i*3]*LH_g_pos[m*3+2];
                                        r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[m*3] + b[i*3+1]*LH_g_pos[m*3+1] + c[i*3+1]*LH_g_pos[m*3+2];
                                        r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[m*3] + b[i*3+2]*LH_g_pos[m*3+1] + c[i*3+2]*LH_g_pos[m*3+2];

                                        if (j<n_tail){
                                                for (xi = 0; xi<3; xi++){
                                                        r_t_tmp[xi] = r_t[j*3+xi];
                                                        LH_g_pos_tmp[xi] = LH_g_pos[m*3+xi];
                                                        a_tmp[xi] = a[i*3+xi];
                                                        b_tmp[xi] = b[i*3+xi];
                                                        c_tmp[xi] = c[i*3+xi];
                                                }
                                                Force_Ele_Vdw_cuda(LH_q[m], t_q[j], 1/(4*PI*k_e), debye,k_ex, LH_vdw_ht[m], r_lho, r_t_tmp,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

                                                force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                                force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                                force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

                                                torque_due_to_force_relative_cuda(force_projection1, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque_hgl);
                                                torque_m[i*2*maxrtlh*3+j*3] += torque_hgl[0];
                                                torque_m[i*2*maxrtlh*3+j*3+1] += torque_hgl[1];
                                                torque_m[i*2*maxrtlh*3+j*3+2] += torque_hgl[2];


                                                if (t_fix[j]==0){
                                                        t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                                        t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                                        t_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];
                                                }else{
                                                        k=0;
                                                        for (int ccnt=0; ccnt<n; ccnt++){
                                                                if (k==(t_grp[j]-1)/10 and type[ccnt]==1){
                                                                        index_c = ccnt;
                                                                        break;
                                                                }
                                                                if(type[ccnt]==1){
                                                                        k=k+1;
                                                                }
                                                        }

                                                        for (int ix = 0; ix<3; ix++){
                                                                r_tmp[ix] = r[index_c*3+ix];
                                                                a_tmp[ix] = a[index_c*3+ix];
                                                                b_tmp[ix] = b[index_c*3+ix];
                                                                c_tmp[ix] = c[index_c*3+ix];
                                                        }
                                                        torque_due_to_force_cuda(force_projection2, r_t_tmp, r_tmp, a_tmp, b_tmp, c_tmp, torque2);
							temp_force_m[i*2*maxrtlh*3+j*3] += force_projection2[0];
			                                temp_force_m[i*2*maxrtlh*3+j*3+1] += force_projection2[1];
                        			        temp_force_m[i*2*maxrtlh*3+j*3+2] += force_projection2[2];
			                                temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3] += torque2[0];
                        			        temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+1] += torque2[1];
			                                temp_force_m[i*2*maxrtlh*3+maxrtlh*3+j*3+2] += torque2[2];
						}
                                        }
                                }
                        }
                }
        }

	__threadfence();
	//Electrostatic and Excluded Volum between LH global head and Linker DNA

        if (i<n){
                if (type[i]==1){
			k=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k=k+1;
                                }
                        }
			if (nc_lh_flag[k]==1){
				for (l=0;l<n_lh_g;l++){
                                        r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[l*3] + b[i*3]*LH_g_pos[l*3+1] + c[i*3]*LH_g_pos[l*3+2];
                                        r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[l*3] + b[i*3+1]*LH_g_pos[l*3+1] + c[i*3+1]*LH_g_pos[l*3+2];
                                        r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[l*3] + b[i*3+2]*LH_g_pos[l*3+1] + c[i*3+2]*LH_g_pos[l*3+2];
                                        if (j<n){
                                                if (type[j]==0){
                                                        for (xi = 0; xi<3; xi++){
                                                                r_tmp[xi] = r[j*3+xi];
                                                                LH_g_pos_tmp[xi] = LH_g_pos[l*3+xi];
                                                                a_tmp[xi] = a[i*3+xi];
                                                                b_tmp[xi] = b[i*3+xi];
                                                                c_tmp[xi] = c[i*3+xi];
                                                        }
                                                        Force_Ele_Vdw_cuda(LH_q[l], q_l, 1/(4*PI*k_e), debye, k_ex, LH_vdw_hl[l], r_lho, r_tmp,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

                                                        force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                                        force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                                        force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                                        force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                                                        torque_due_to_force_relative_cuda(force_projection1, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque_hgl);
                                                        torque_m[i*2*maxrtlh*3+j*3] += torque_hgl[0];
                                                        torque_m[i*2*maxrtlh*3+j*3+1] += torque_hgl[1];
                                                        torque_m[i*2*maxrtlh*3+j*3+2] += torque_hgl[2];

                                                }
                                        }
                                }
                        }
                }
        }

	__threadfence();

	//Electrostatic and Excluded Volum between LH global head and core

        if (i<n){
                if (type[i]==1){
			k=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k=k+1;
                                }
                        }
			if (nc_lh_flag[k]==1){
                                for (l=0;l<n_lh_g;l++){
                                        r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[l*3] + b[i*3]*LH_g_pos[l*3+1] + c[i*3]*LH_g_pos[l*3+2];
                                        r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[l*3] + b[i*3+1]*LH_g_pos[l*3+1] + c[i*3+1]*LH_g_pos[l*3+2];
                                        r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[l*3] + b[i*3+2]*LH_g_pos[l*3+1] + c[i*3+2]*LH_g_pos[l*3+2];
                                        if (j<n){
                                                if (i!=j and type[j]==1){
                                                        for (m=0; m<Nq; m++){
                                                                z[0] = r[j*3]+a[j*3]*core_pos[m*3]+b[j*3]*core_pos[m*3+1]+c[j*3]*core_pos[m*3+2];
                                                                z[1] = r[j*3+1]+a[j*3+1]*core_pos[m*3]+b[j*3+1]*core_pos[m*3+1]+c[j*3+1]*core_pos[m*3+2];
                                                                z[2] = r[j*3+2]+a[j*3+2]*core_pos[m*3]+b[j*3+2]*core_pos[m*3+1]+c[j*3+2]*core_pos[m*3+2];

                                                                Force_Ele_Vdw_cuda(LH_q[l], core_q[m], 1/(4*PI*k_e), debye, k_ex, LH_vdw_hc[l], r_lho, z, force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

                                                                for (xi = 0; xi<3; xi++){
                                                                        LH_g_pos_tmp[xi] = LH_g_pos[l*3+xi];
                                                                        a_tmp[xi] = a[i*3+xi];
                                                                        b_tmp[xi] = b[i*3+xi];
                                                                        c_tmp[xi] = c[i*3+xi];
                                                                }

                                                                force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                                                force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                                                force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
                                                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                                                                torque_due_to_force_relative_cuda(force_projection1, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque1);
                                                                torque_m[i*2*maxrtlh*3+j*3] += torque1[0];
                                                                torque_m[i*2*maxrtlh*3+j*3+1] += torque1[1];
                                                                torque_m[i*2*maxrtlh*3+j*3+2] += torque1[2];
                                                                for (xi = 0; xi<3; xi++){
                                                                        core_pos_tmp[xi] = core_pos[m*3+xi];
                                                                        a_tmp[xi] = a[j*3+xi];
                                                                        b_tmp[xi] = b[j*3+xi];
                                                                        c_tmp[xi] = c[j*3+xi];
                                                                }

                                                                torque_due_to_force_relative_cuda(force_projection2, core_pos_tmp, a_tmp, b_tmp, c_tmp, torque2);
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += torque2[0];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += torque2[1];
                                                                torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += torque2[2];
                                                        }
                                                }
                                        }
                                }
                        }
                }
        }

	__threadfence();

	//Electrostatic and Excluded Volume between LH global head and LH


        if (i<n){
                if (type[i]==1){
			k2=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k2=k2+1;
                                }
                        }
                        if (nc_lh_flag[k2]==1){
				count = 0;
                                for (int cntc=0;cntc<k2;cntc++){
                                        if (nc_lh_flag[cntc]==1){
                                                count = count+1;
                                        }
                                }
                                for (m=0;m<n_lh_g;m++){
                                        r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[m*3] + b[i*3]*LH_g_pos[m*3+1] + c[i*3]*LH_g_pos[m*3+2];
                                        r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[m*3] + b[i*3+1]*LH_g_pos[m*3+1] + c[i*3+1]*LH_g_pos[m*3+2];
                                        r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[m*3] + b[i*3+2]*LH_g_pos[m*3+1] + c[i*3+2]*LH_g_pos[m*3+2];

                                        if (j<n_LH){
                                                if (j/(n_lh_n +n_lh_c) != count){
                                                        index2 = j%(n_lh_n+n_lh_c);
                                                        if (index2 < n_lh_n){
                                                                index2 = index2;
                                                        }else{
                                                                index2 = index2 + n_lh_g;
                                                        }

                                                        for (xi = 0; xi<3; xi++){
                                                                r_lh_tmp2[xi] = r_lh[j*3+xi];
                                                                LH_g_pos_tmp[xi] = LH_g_pos[m*3+xi];
                                                                a_tmp[xi] = a[i*3+xi];
                                                                b_tmp[xi] = b[i*3+xi];
                                                                c_tmp[xi] = c[i*3+xi];
                                                        }

                                                        Force_Ele_Vdw_cuda(LH_q[m], LH_q[index2], 1/(4*PI*k_e), debye,k_ex, (LH_vdw_hh[m]+LH_vdw_hh[index2])/2, r_lho, r_lh_tmp2,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

							force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                                                        force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
                                                        force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

							LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
                			                LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
		                        	        LH_force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                                                        torque_due_to_force_relative_cuda(force_projection1, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque_hgl);
                                                        torque_m[i*2*maxrtlh*3+j*3] += torque_hgl[0];
                                                        torque_m[i*2*maxrtlh*3+j*3+1] += torque_hgl[1];
                                                        torque_m[i*2*maxrtlh*3+j*3+2] += torque_hgl[2];

                                                }
                                        }
                                }
                                
                        }
                }
        }

	__threadfence();

	//Electrostatic and Excluded Volume between LH global head and LH global head

        if (i<n){
                if (type[i]==1){
			k2=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k2=k2+1;
                                }
                        }
                        if (nc_lh_flag[k2]==1){
                                for (m=0;m<n_lh_g;m++){
                                        r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[m*3] + b[i*3]*LH_g_pos[m*3+1] + c[i*3]*LH_g_pos[m*3+2];
                                        r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[m*3] + b[i*3+1]*LH_g_pos[m*3+1] + c[i*3+1]*LH_g_pos[m*3+2];
                                        r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[m*3] + b[i*3+2]*LH_g_pos[m*3+1] + c[i*3+2]*LH_g_pos[m*3+2];

                                        k = k2+1;
                                        if (j>=i+1 and j<n){
                                                if (type[j]==1){
                                                        if (nc_lh_flag[k]==1){
                                                                for (m1=0;m1<n_lh_g;m1++){
                                                                        r_lho2[0] = r[j*3] + a[j*3]*LH_g_pos[m1*3] + b[j*3]*LH_g_pos[m1*3+1] + c[j*3]*LH_g_pos[m1*3+2];
                                                                        r_lho2[1] = r[j*3+1] + a[j*3+1]*LH_g_pos[m1*3] + b[j*3+1]*LH_g_pos[m1*3+1] + c[j*3+1]*LH_g_pos[m1*3+2];
                                                                        r_lho2[2] = r[j*3+2] + a[j*3+2]*LH_g_pos[m1*3] + b[j*3+2]*LH_g_pos[m1*3+1] + c[j*3+2]*LH_g_pos[m1*3+2];

                                                                        Force_Ele_Vdw_cuda(LH_q[m], LH_q[m1], 1/(4*PI*k_e), debye,k_ex, (LH_vdw_hh[m]+LH_vdw_hh[m1])/2, r_lho, r_lho2,force_projection1, force_projection2, Energy_m[i*maxrtlh+j]);

									force_m[i*2*maxrtlh*3+j*3] += force_projection1[0];
                		                                        force_m[i*2*maxrtlh*3+j*3+1] += force_projection1[1];
		                                                        force_m[i*2*maxrtlh*3+j*3+2] += force_projection1[2];

									force_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += force_projection2[0];
					                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += force_projection2[1];
					                                force_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += force_projection2[2];

                                                                        for (xi = 0; xi<3; xi++){
                                                                                LH_g_pos_tmp[xi] = LH_g_pos[m*3+xi];
                                                                                a_tmp[xi] = a[i*3+xi];
                                                                                b_tmp[xi] = b[i*3+xi];
                                                                                c_tmp[xi] = c[i*3+xi];
                                                                        }

                                                                        torque_due_to_force_relative_cuda(force_projection1, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque_hgl);
                                                                        torque_m[i*2*maxrtlh*3+j*3] += torque_hgl[0];
                                                                        torque_m[i*2*maxrtlh*3+j*3+1] += torque_hgl[1];
                                                                        torque_m[i*2*maxrtlh*3+j*3+2] += torque_hgl[2];

                                                                        for (xi = 0; xi<3; xi++){
                                                                                LH_g_pos_tmp[xi] = LH_g_pos[m1*3+xi];
                                                                                a_tmp[xi] = a[j*3+xi];
                                                                                b_tmp[xi] = b[j*3+xi];
                                                                                c_tmp[xi] = c[j*3+xi];
                                                                        }

                                                                        torque_due_to_force_relative_cuda(force_projection2, LH_g_pos_tmp, a_tmp, b_tmp, c_tmp, torque_hgl);
                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3] += torque_hgl[0];
                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+1] += torque_hgl[1];
                                                                        torque_m[j*2*maxrtlh*3+maxrtlh*3+i*3+2] += torque_hgl[2];

                                                                }
                                                        }
                                                }
                                        }
                                }

                        }
                        
                }
        }

	__threadfence();

	//Stretching between LH global head with C-term


        if (i < n){
                if (type[i]==1){
			k2=0;
                        for (int cnti=0; cnti<i;cnti++){
                                if (type[cnti]==1){
                                        k2=k2+1;
                                }
                        }
                        if (nc_lh_flag[k2]==1){
				k = 0;
                                for (int cntc=0;cntc<k2;cntc++){
                                        if (nc_lh_flag[cntc]==1){
                                                k = k+1;
                                        }
                                }
                                if (j < n_lh_g){
                                        if (LH_conn[j+n_lh_n]==1){
                                                r_lho[0] = r[i*3] + a[i*3]*LH_g_pos[j*3] + b[i*3]*LH_g_pos[j*3+1] + c[i*3]*LH_g_pos[j*3+2];
                                                r_lho[1] = r[i*3+1] + a[i*3+1]*LH_g_pos[j*3] + b[i*3+1]*LH_g_pos[j*3+1] + c[i*3+1]*LH_g_pos[j*3+2];
                                                r_lho[2] = r[i*3+2] + a[i*3+2]*LH_g_pos[j*3] + b[i*3+2]*LH_g_pos[j*3+1] + c[i*3+2]*LH_g_pos[j*3+2];

                                                distance = (r_lh[k*nlh*3]-r_lho[0])*(r_lh[k*nlh*3]-r_lho[0])+(r_lh[k*nlh*3+1]-r_lho[1])*(r_lh[k*nlh*3+1]-r_lho[1])+(r_lh[k*nlh*3+2]-r_lho[2])*(r_lh[k*nlh*3+2]-r_lho[2]);
						distance = sqrt(distance);

                                                Energy_m[i*maxrtlh+j] = Energy_m[i*maxrtlh+j] + LH_kstr[j+n_lh_n]*(distance-LH_streq[j+n_lh_n])*(distance-LH_streq[j+n_lh_n])/2;


                                                Stri[0] = (distance - LH_streq[j+n_lh_n])*(r_lh[k*nlh*3]-r_lho[0])/distance;
                                                Stri[1] = (distance - LH_streq[j+n_lh_n])*(r_lh[k*nlh*3+1]-r_lho[1])/distance;
                                                Stri[2] = (distance - LH_streq[j+n_lh_n])*(r_lh[k*nlh*3+2]-r_lho[2])/distance;


                                                LH_force_m[k*nlh*2*maxrtlh*3+j*3] -= LH_kstr[j+n_lh_n]*Stri[0]*10;
                                                LH_force_m[k*nlh*2*maxrtlh*3+j*3+1] -= LH_kstr[j+n_lh_n]*Stri[1]*10;
                                                LH_force_m[k*nlh*2*maxrtlh*3+j*3+2] -= LH_kstr[j+n_lh_n]*Stri[2]*10;

                                                force_m[i*2*maxrtlh*3+j*3] += LH_kstr[j+n_lh_n]*Stri[0]*10;
                                                force_m[i*2*maxrtlh*3+j*3+1] += LH_kstr[j+n_lh_n]*Stri[1]*10;
                                                force_m[i*2*maxrtlh*3+j*3+2] += LH_kstr[j+n_lh_n]*Stri[2]*10;

                                                fa = LH_kstr[j+n_lh_n]*(a[i*3]*Stri[0]+a[i*3+1]*Stri[1]+a[i*3+2]*Stri[2]);
                                                fb = LH_kstr[j+n_lh_n]*(b[i*3]*Stri[0]+b[i*3+1]*Stri[1]+b[i*3+2]*Stri[2]);
                                                fc = LH_kstr[j+n_lh_n]*(c[i*3]*Stri[0]+c[i*3+1]*Stri[1]+c[i*3+2]*Stri[2]);
                                                torque_m[i*2*maxrtlh*3+j*3] += fc*LH_g_pos[j*3+1] - fb*LH_g_pos[j*3+2];
                                                torque_m[i*2*maxrtlh*3+j*3+1] += fa*LH_g_pos[j*3+2] - fc*LH_g_pos[j*3];
                                                torque_m[i*2*maxrtlh*3+j*3+2] += fb*LH_g_pos[j*3] - fa*LH_g_pos[j*3+1];

                                        }
                                }
                        }
                }
        }

}

__global__ void force_torque_tmp(int n3, int n_tail3, int n_LH3, double* force, double* torque, double* t_force, double* LH_force, double* force_n, double* torque_n, double* t_force_n, double* LH_force_n, double* force_tmp, double* torque_tmp, double* t_force_tmp, double* LH_force_tmp){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n3){
		force_tmp[i] = 0.5*(force[i]+force_n[i]);
		torque_tmp[i] = 0.5*(torque[i]+torque_n[i]);
	}
	if (i<n_tail3){
		t_force_tmp[i] = 0.5*(t_force[i]+t_force_n[i]);
	}
	if (i<n_LH3){
		LH_force_tmp[i] = 0.5*(LH_force[i]+LH_force_n[i]);
	}

}

__global__ void final_updates(int n3, int n_tail3, int n_LH3, double* r, double* r_n, double* a, double* a_n, double* b, double* b_n, double* c, double* c_n, double* r_t, double* r_t_n, double* r_lh, double* r_lh_n){

	int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < n3){
		r[i] = r_n[i];
		a[i] = a_n[i];
		b[i] = b_n[i];
		c[i] = c_n[i];
	}

	if(i < n_tail3){
		r_t[i] = r_t_n[i];
	}

	if(i < n_LH3){
		r_lh[i] = r_lh_n[i];
	}

}


extern "C++" void cuda_application_init_D_Chol(int n3){


	size_t bytes_D = n3*n3*sizeof(double);
        size_t bytes_Chol = n3*n3*sizeof(double);

	hipMalloc(&d_D, bytes_D);
        hipMalloc(&d_Chol, bytes_Chol);

}

extern "C++" void cuda_application_init_data(int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p, double h, double g, double s, double* phi_o, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, int Nq, int Nq3, double* core_pos, double* core_q, int n_t, int n_tail, int n_tail3, double* tail_pos, int* tail_fix, int* nc_t_flag, double* r_t, double* beta_t, double* h_t, double* g_t, double* lo_t, double* beta_o_t, double* t_q, double* t_rad, int* t_grp, int* t_fix, int n_lh_n, int n_lh_g, int n_lh_c, int n_LH, int n_LH3, double* LH_g_pos, int* LH_conn, int* nc_lh_flag, double* beta_lh, double* r_lh, double* LH_q, double* LH_vdw_hh, double* LH_vdw_hc, double* LH_vdw_hl, double* LH_vdw_ht, double* LH_kstr, double* LH_kben, double* LH_streq, double* LH_betaeq, double* LH_force, double* t_force, double* force, double* torque, double* Energy, double* r_all, double* rad_all){


	int n_D = n+n_tail+n_LH;
	int n_D3 = n_D*3;

	size_t bytes = sizeof(double);
        size_t bytes_ni = n*sizeof(int);
        size_t bytes_nd = n*sizeof(double);
        size_t bytes_n3 = n3*sizeof(double);
        size_t bytes_nc = n_c*sizeof(double);
        size_t bytes_nc3 = nc3*sizeof(double);
        size_t bytes_Nq = Nq*sizeof(double);
        size_t bytes_Nq3 = Nq3*sizeof(double);

        size_t bytes_t = n_t*sizeof(int);
        size_t bytes_t3 = n_t*3*sizeof(double);
        size_t bytes_tail = n_tail*sizeof(double);
        size_t bytes_tail3 = n_tail3*sizeof(double);
        size_t bytes_tail_int = n_tail*sizeof(int);
        size_t bytes_nc_int = n_c*sizeof(int);

        size_t bytes_lh_g = n_lh_g*3*sizeof(double);
        size_t bytes_lh_conn = (n_lh_g+n_lh_c+n_lh_n)*sizeof(int);
        size_t bytes_LH = n_LH*sizeof(double);
        size_t bytes_LH3 = n_LH3*sizeof(double);
        size_t bytes_lh_d = (n_lh_g+n_lh_c+n_lh_n)*sizeof(double);

	size_t bytes_n_D = n_D*sizeof(double);
	size_t bytes_n_D3 = n_D3*sizeof(double);

        int maxrtlh;

        if (n>n_LH){
                if (n>n_tail){
                        maxrtlh=n;
                }else{
                        maxrtlh=n_tail;
                }
        }else{
                if (n_LH>n_tail){
                        maxrtlh=n_LH;
                }else{
                        maxrtlh=n_tail;
                }
        }

        size_t bytes_r_t_lh_m = maxrtlh*maxrtlh*6*sizeof(double);

	size_t bytes_r_t_lh_m_E = maxrtlh*maxrtlh*sizeof(double);

	//Allocate memory on device side

        hipMalloc(&d_Energy, bytes);
        hipMalloc(&d_type, bytes_ni);
        hipMalloc(&d_r, bytes_n3);
        hipMalloc(&d_a, bytes_n3);
        hipMalloc(&d_b, bytes_n3);
        hipMalloc(&d_c, bytes_n3);
        hipMalloc(&d_alpha, bytes_nd);
        hipMalloc(&d_beta, bytes_nd);
        hipMalloc(&d_gamma, bytes_nd);
        hipMalloc(&d_length, bytes_nd);
        hipMalloc(&d_a_dna, bytes_nc3);
        hipMalloc(&d_b_dna, bytes_nc3);
        hipMalloc(&d_c_dna, bytes_nc3);
        hipMalloc(&d_alpha_p, bytes_nc);
        hipMalloc(&d_beta_p, bytes_nc);
        hipMalloc(&d_gamma_p, bytes_nc);
        hipMalloc(&d_phi_o, bytes_nd);
        hipMalloc(&d_core_pos, bytes_Nq3);
        hipMalloc(&d_core_q, bytes_Nq);
        hipMalloc(&d_force, bytes_n3);
        hipMalloc(&d_torque, bytes_n3);
	hipMalloc(&d_force_n, bytes_n3);
        hipMalloc(&d_torque_n, bytes_n3);

	hipMalloc(&d_force_tmp, bytes_n3);
        hipMalloc(&d_torque_tmp, bytes_n3);

	hipMalloc(&d_r_all, bytes_n_D3);
	hipMalloc(&d_rad_all, bytes_n_D);
	hipMalloc(&d_d_theta, bytes_n3);
	hipMalloc(&d_rd, bytes_n_D3);

	hipMalloc(&d_r_n, bytes_n3);
        hipMalloc(&d_a_n, bytes_n3);
        hipMalloc(&d_b_n, bytes_n3);
        hipMalloc(&d_c_n, bytes_n3);
        hipMalloc(&d_alpha_n, bytes_nd);
        hipMalloc(&d_beta_n, bytes_nd);
        hipMalloc(&d_gamma_n, bytes_nd);
        hipMalloc(&d_length_n, bytes_nd);
        hipMalloc(&d_a_dna_n, bytes_nc3);
        hipMalloc(&d_b_dna_n, bytes_nc3);
        hipMalloc(&d_c_dna_n, bytes_nc3);
        hipMalloc(&d_alpha_p_n, bytes_nc);
        hipMalloc(&d_beta_p_n, bytes_nc);
        hipMalloc(&d_gamma_p_n, bytes_nc);

        hipMalloc(&d_tail_pos, bytes_t3);
        hipMalloc(&d_tail_fix, bytes_t);
        hipMalloc(&d_nc_t_flag, bytes_nc_int);
        hipMalloc(&d_r_t, bytes_tail3);
        hipMalloc(&d_beta_t, bytes_tail);
        hipMalloc(&d_h_t, bytes_tail);
        hipMalloc(&d_g_t, bytes_tail);
        hipMalloc(&d_lo_t, bytes_tail);
        hipMalloc(&d_beta_o_t, bytes_tail);
        hipMalloc(&d_t_q, bytes_tail);
        hipMalloc(&d_t_rad, bytes_tail);
        hipMalloc(&d_t_grp, bytes_tail_int);
        hipMalloc(&d_t_fix, bytes_tail_int);
        hipMalloc(&d_t_force, bytes_tail3);
	hipMalloc(&d_t_force_n, bytes_tail3);

	hipMalloc(&d_t_force_tmp, bytes_tail3);

	hipMalloc(&d_r_t_n, bytes_tail3);
        hipMalloc(&d_beta_t_n, bytes_tail);

        hipMalloc(&d_LH_g_pos, bytes_lh_g);
        hipMalloc(&d_LH_conn, bytes_lh_conn);
        hipMalloc(&d_nc_lh_flag, bytes_nc_int);
        hipMalloc(&d_beta_lh, bytes_LH);
        hipMalloc(&d_r_lh, bytes_LH3);
        hipMalloc(&d_LH_q, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hh, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hc, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hl, bytes_lh_d);
        hipMalloc(&d_LH_vdw_ht, bytes_lh_d);
        hipMalloc(&d_LH_kstr, bytes_lh_d);
        hipMalloc(&d_LH_kben, bytes_lh_d);
        hipMalloc(&d_LH_streq, bytes_lh_d);
        hipMalloc(&d_LH_betaeq, bytes_lh_d);
        hipMalloc(&d_LH_force, bytes_LH3);
	hipMalloc(&d_LH_force_n, bytes_LH3);

	hipMalloc(&d_LH_force_tmp, bytes_LH3);

	hipMalloc(&d_beta_lh_n, bytes_LH);
        hipMalloc(&d_r_lh_n, bytes_LH3);

        hipMalloc(&d_force_m, bytes_r_t_lh_m);
        hipMalloc(&d_torque_m, bytes_r_t_lh_m);
        hipMalloc(&d_t_force_m, bytes_r_t_lh_m);
        hipMalloc(&d_LH_force_m, bytes_r_t_lh_m);
        hipMalloc(&d_temp_force_m, bytes_r_t_lh_m);

	hipMalloc(&d_Energy_m, bytes_r_t_lh_m_E);

	//Copy data to the device

        hipMemcpy(d_Energy, Energy, bytes,hipMemcpyHostToDevice);
        hipMemcpy(d_type, type, bytes_ni, hipMemcpyHostToDevice);
        hipMemcpy(d_r, r, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_a, a, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_c, c, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_alpha, alpha, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_beta, beta, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_gamma, gamma, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_length, length, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_a_dna, a_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_b_dna, b_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_c_dna, c_dna, bytes_nc3, hipMemcpyHostToDevice);
        hipMemcpy(d_alpha_p, alpha_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_p, beta_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_gamma_p, gamma_p, bytes_nc, hipMemcpyHostToDevice);
        hipMemcpy(d_phi_o, phi_o, bytes_nd, hipMemcpyHostToDevice);
        hipMemcpy(d_core_pos, core_pos, bytes_Nq3, hipMemcpyHostToDevice);
        hipMemcpy(d_core_q, core_q, bytes_Nq, hipMemcpyHostToDevice);
        hipMemcpy(d_force, force, bytes_n3, hipMemcpyHostToDevice);
        hipMemcpy(d_torque, torque, bytes_n3, hipMemcpyHostToDevice);

	hipMemcpy(d_r_all, r_all, bytes_n_D3, hipMemcpyHostToDevice);
	hipMemcpy(d_rad_all, rad_all, bytes_n_D, hipMemcpyHostToDevice);

        hipMemcpy(d_tail_pos, tail_pos, bytes_t3, hipMemcpyHostToDevice);
        hipMemcpy(d_tail_fix, tail_fix, bytes_t, hipMemcpyHostToDevice);
        hipMemcpy(d_nc_t_flag, nc_t_flag, bytes_nc_int, hipMemcpyHostToDevice);
        hipMemcpy(d_r_t, r_t, bytes_tail3, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_t, beta_t, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_h_t, h_t, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_g_t, g_t, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_lo_t, lo_t, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_o_t, beta_o_t, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_t_q, t_q, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_t_rad, t_rad, bytes_tail, hipMemcpyHostToDevice);
        hipMemcpy(d_t_grp, t_grp, bytes_tail_int, hipMemcpyHostToDevice);
        hipMemcpy(d_t_fix, t_fix, bytes_tail_int, hipMemcpyHostToDevice);
        hipMemcpy(d_t_force, t_force, bytes_tail3, hipMemcpyHostToDevice);

        hipMemcpy(d_LH_g_pos, LH_g_pos, bytes_lh_g, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_conn, LH_conn, bytes_lh_conn, hipMemcpyHostToDevice);
        hipMemcpy(d_nc_lh_flag, nc_lh_flag, bytes_nc_int, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_lh, beta_lh, bytes_LH, hipMemcpyHostToDevice);
        hipMemcpy(d_r_lh, r_lh, bytes_LH3, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_q, LH_q, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hh, LH_vdw_hh, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hc, LH_vdw_hc, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hl, LH_vdw_hl, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_ht, LH_vdw_ht, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_kstr, LH_kstr, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_kben, LH_kben, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_streq, LH_streq, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_betaeq, LH_betaeq, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_force, LH_force, bytes_LH3, hipMemcpyHostToDevice);



}
/*
extern "C++" void cuda_application_D_Chol(int n, int n3, double* r, double a1, double a2, double* rad){


	//Free D and Chol
	hipFree(d_D);
	hipFree(d_Chol);

	//initialize cuSolver and cublas
	hipsolverHandle_t solver_handle;
        hipsolverDnCreate(&solver_handle);

        int work_size = 0;

        int *devInfo;
        hipMalloc(&devInfo, sizeof(int));

	//Device Matrices

        double* d_r;
        double* d_rad;

        size_t bytes = n3*sizeof(double);
        size_t bytes_D = n3*n3*sizeof(double);
        size_t bytes_Chol = n3*n3*sizeof(double);

	//Allocate memory on device side

        hipMalloc(&d_r, bytes);
        hipMalloc(&d_rad, bytes);

        hipMalloc(&d_D, bytes_D);
        hipMalloc(&d_Chol, bytes_Chol);

	//Copy data to the device

        hipMemcpy(d_r, r, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_rad, rad, bytes, hipMemcpyHostToDevice);

	//assign number of blocks and threads

	int threads_per_block = 32;
        dim3 block_size(threads_per_block, threads_per_block);
        dim3 grid_size(n3 / block_size.x + 1 , n3 / block_size.y + 1 );

	//Calculate D

	Diffusion_Tensor_CUDA <<<grid_size, block_size>>> (n, n3, d_r, a1, a2, d_rad, d_D, d_Chol);

	//Calculate Chol

	hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_UPPER, n3, d_Chol, n3, &work_size);

	double *work;
        hipMalloc(&work, work_size * sizeof(double));

        hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, n3, d_Chol, n3, work, work_size, devInfo);

        Cholesky_Decomposition_mod <<<grid_size, block_size>>> (n3, d_Chol);

	hipFree(d_r);
        hipFree(d_rad);

	hipFree(devInfo);
        hipFree(work);

	hipsolverDnDestroy(solver_handle);

}
*/
/*
extern "C++" void cuda_application_rd(int n, int n3, double* r, double a1, double a2, double* rad, double s2dt, double* p, double* rd){

	//Device Matrices

	double* d_r;
	double* d_rad; 
	double* d_rd;
	double* d_p;

	size_t bytes = n3*sizeof(double);

	//Allocate memory on device side

	hipMalloc(&d_r, bytes);
	hipMalloc(&d_rad, bytes);
	hipMalloc(&d_rd, bytes);
	hipMalloc(&d_p, bytes);

	//Copy data to the device

	hipMemcpy(d_r, r, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_rad, rad, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

	//assign number of blocks and threads

	int num_thread = 32;
	int num_block = (n3 + num_thread -1) / num_thread;

	//calculate rd

	rd_cal <<<num_block, num_thread>>> (n3, d_rd, d_Chol, d_p, s2dt);

	//copy rd back to the host

	hipMemcpy(rd, d_rd, bytes, hipMemcpyDeviceToHost);

	hipFree(d_r);
	hipFree(d_rad);
	hipFree(d_rd);
	hipFree(d_p);

}
*/
/*
extern "C++" void cuda_application_translate(int n, int n_D3, int n3, double* r_all, double* r, double a1, double a2, double* rad, double del, double* force_global, double* t_force_global, double* LH_force_global, int n_tail3, int n_LH3, double* rd, double* r_t, double* r_lh, double* r_n, double* r_t_n, double* r_lh_n){

        //Device Matrices

        double* d_rall;
        double* d_rad;

	double* d_r;
	double* d_r_n;
	double* d_rd;
	double* d_r_t;
	double* d_r_t_n;
	double* d_r_lh;
	double* d_r_lh_n;
	double* d_force_global;
	double* d_t_force_global;
	double* d_LH_force_global;


        size_t bytes = n_D3*sizeof(double);
	size_t bytes_n = n3*sizeof(double);
	size_t bytes_t = n_tail3*sizeof(double);
	size_t bytes_LH = n_LH3*sizeof(double);



        //Allocate memory on device side

        hipMalloc(&d_rall, bytes);
        hipMalloc(&d_rad, bytes);

	hipMalloc(&d_r, bytes_n);
	hipMalloc(&d_r_n, bytes_n);
	hipMalloc(&d_rd, bytes);
	hipMalloc(&d_r_t, bytes_t);
	hipMalloc(&d_r_t_n, bytes_t);
	hipMalloc(&d_r_lh, bytes_LH);
	hipMalloc(&d_r_lh_n, bytes_LH);
	hipMalloc(&d_force_global, bytes_n);
	hipMalloc(&d_t_force_global, bytes_t);
	hipMalloc(&d_LH_force_global, bytes_LH);


        //Copy data to the device

        hipMemcpy(d_rall, r_all, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_rad, rad, bytes, hipMemcpyHostToDevice);

	hipMemcpy(d_r, r, bytes_n, hipMemcpyHostToDevice);
	hipMemcpy(d_rd, rd, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_r_t, r_t, bytes_t, hipMemcpyHostToDevice);
	hipMemcpy(d_r_lh, r_lh, bytes_LH, hipMemcpyHostToDevice);
	hipMemcpy(d_force_global, force_global, bytes_n, hipMemcpyHostToDevice);
	hipMemcpy(d_t_force_global, t_force_global, bytes_t, hipMemcpyHostToDevice);
	hipMemcpy(d_LH_force_global, LH_force_global, bytes_LH, hipMemcpyHostToDevice);

	int num_thread = 32;
        int num_block = (n_D3 + num_thread -1) / num_thread;

	translation_cal <<<num_block, num_thread>>> (n_D3, n3, n_tail3, n_LH3, d_r, d_r_n, d_rd, d_r_t, d_r_t_n, d_r_lh, d_r_lh_n, del, d_force_global, d_t_force_global, d_LH_force_global, d_D);

	hipMemcpy(r_n, d_r_n, bytes_n, hipMemcpyDeviceToHost);
	hipMemcpy(r_t_n, d_r_t_n, bytes_t, hipMemcpyDeviceToHost);
	hipMemcpy(r_lh_n, d_r_lh_n, bytes_LH, hipMemcpyDeviceToHost);


        hipFree(d_rall);
        hipFree(d_rad);

	hipFree(d_r);
        hipFree(d_r_n);
        hipFree(d_rd);
	hipFree(d_r_t);
        hipFree(d_r_t_n);
        hipFree(d_r_lh);
	hipFree(d_r_lh_n);
        hipFree(d_force_global);
        hipFree(d_t_force_global);
	hipFree(d_LH_force_global);

}
*/
/*
extern "C++" void force_and_torque_cuda_application(int n_c, int nc3, int n, int n3, int* type, double* r, double* a, double* b, double* c, double* alpha, double* beta, double* gamma, double* length, double* a_dna, double* b_dna, double* c_dna, double* alpha_p, double* beta_p, double* gamma_p, double h, double g, double s, double* phi_o, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, int Nq, int Nq3, double* core_pos, double* core_q, int n_t, int n_tail, int n_tail3, double* tail_pos, int* tail_fix, int* nc_t_flag, double* r_t, double* beta_t, double* h_t, double* g_t, double* lo_t, double* beta_o_t, double* t_q, double* t_rad, int* t_grp, int* t_fix, int n_lh_n, int n_lh_g, int n_lh_c, int n_LH, int n_LH3, double* LH_g_pos, int* LH_conn, int* nc_lh_flag, double* beta_lh, double* r_lh, double* LH_q, double* LH_vdw_hh, double* LH_vdw_hc, double* LH_vdw_hl, double* LH_vdw_ht, double* LH_kstr, double* LH_kben, double* LH_streq, double* LH_betaeq, double* LH_force, double* t_force, double* force, double* torque, double* Energy){


	//Device Matrices
	int* d_type;
	double* d_r;
	double* d_a;
	double* d_b;
	double* d_c;
	double* d_alpha;
	double* d_beta;
	double* d_gamma;
	double* d_length;
	double* d_a_dna;
	double* d_b_dna;
	double* d_c_dna;
	double* d_alpha_p;
	double* d_beta_p;
	double* d_gamma_p;
	double* d_phi_o;
	double* d_force;
	double* d_torque;
	double* d_Energy;
	double* d_core_pos;
	double* d_core_q;

	double* d_tail_pos;
	int* d_tail_fix;
	int* d_nc_t_flag;
	double* d_r_t;
	double* d_beta_t;
	double* d_h_t;
	double* d_g_t;
	double* d_lo_t;
	double* d_beta_o_t;
	double* d_t_q;
	double* d_t_rad;
	int* d_t_grp;
	int* d_t_fix;
	double* d_t_force;

	double* d_LH_g_pos; 
	int* d_LH_conn; 
	int* d_nc_lh_flag; 
	double* d_beta_lh; 
	double* d_r_lh; 
	double* d_LH_q; 
	double* d_LH_vdw_hh; 
	double* d_LH_vdw_hc; 
	double* d_LH_vdw_hl; 
	double* d_LH_vdw_ht; 
	double* d_LH_kstr; 
	double* d_LH_kben; 
	double* d_LH_streq; 
	double* d_LH_betaeq; 
	double* d_LH_force; 

	double* d_force_m;
	double* d_torque_m;
	double* d_t_force_m;
	double* d_LH_force_m;

	double* d_temp_force_m;

	size_t bytes = sizeof(double);
	size_t bytes_ni = n*sizeof(int);
	size_t bytes_nd = n*sizeof(double);
	size_t bytes_n3 = n3*sizeof(double);
	size_t bytes_nc = n_c*sizeof(double);
	size_t bytes_nc3 = nc3*sizeof(double);
	size_t bytes_Nq = Nq*sizeof(double);
	size_t bytes_Nq3 = Nq3*sizeof(double);
	
	size_t bytes_t = n_t*sizeof(int);
	size_t bytes_t3 = n_t*3*sizeof(double);
	size_t bytes_tail = n_tail*sizeof(double);
	size_t bytes_tail3 = n_tail3*sizeof(double);
	size_t bytes_tail_int = n_tail*sizeof(int);
	size_t bytes_nc_int = n_c*sizeof(int);

	size_t bytes_lh_g = n_lh_g*3*sizeof(double);
	size_t bytes_lh_conn = (n_lh_g+n_lh_c+n_lh_n)*sizeof(int);
	size_t bytes_LH = n_LH*sizeof(double);
	size_t bytes_LH3 = n_LH3*sizeof(double);
	size_t bytes_lh_d = (n_lh_g+n_lh_c+n_lh_n)*sizeof(double);
	
	int maxrtlh;

	if (n>n_LH){
		if (n>n_tail){
			maxrtlh=n;
		}else{
			maxrtlh=n_tail;
		}
	}else{
		if (n_LH>n_tail){
			maxrtlh=n_LH;
		}else{
			maxrtlh=n_tail;
		}
	}

	size_t bytes_r_t_lh_m = maxrtlh*maxrtlh*6*sizeof(double);

	//Allocate memory on device side

	hipMalloc(&d_Energy, bytes);
        hipMalloc(&d_type, bytes_ni);
	hipMalloc(&d_r, bytes_n3);
	hipMalloc(&d_a, bytes_n3);
	hipMalloc(&d_b, bytes_n3);
	hipMalloc(&d_c, bytes_n3);
	hipMalloc(&d_alpha, bytes_nd);
	hipMalloc(&d_beta, bytes_nd);
	hipMalloc(&d_gamma, bytes_nd);
	hipMalloc(&d_length, bytes_nd);
	hipMalloc(&d_a_dna, bytes_nc3);
	hipMalloc(&d_b_dna, bytes_nc3);
	hipMalloc(&d_c_dna, bytes_nc3);
	hipMalloc(&d_alpha_p, bytes_nc);
	hipMalloc(&d_beta_p, bytes_nc);
	hipMalloc(&d_gamma_p, bytes_nc);
	hipMalloc(&d_phi_o, bytes_nd);
	hipMalloc(&d_core_pos, bytes_Nq3);
	hipMalloc(&d_core_q, bytes_Nq);
	hipMalloc(&d_force, bytes_n3);
	hipMalloc(&d_torque, bytes_n3);

	hipMalloc(&d_tail_pos, bytes_t3);
	hipMalloc(&d_tail_fix, bytes_t);
	hipMalloc(&d_nc_t_flag, bytes_nc_int);
	hipMalloc(&d_r_t, bytes_tail3);
	hipMalloc(&d_beta_t, bytes_tail);
	hipMalloc(&d_h_t, bytes_tail);
	hipMalloc(&d_g_t, bytes_tail);
	hipMalloc(&d_lo_t, bytes_tail);
	hipMalloc(&d_beta_o_t, bytes_tail);
	hipMalloc(&d_t_q, bytes_tail);
	hipMalloc(&d_t_rad, bytes_tail);
	hipMalloc(&d_t_grp, bytes_tail_int);
	hipMalloc(&d_t_fix, bytes_tail_int);
	hipMalloc(&d_t_force, bytes_tail3);

	hipMalloc(&d_LH_g_pos, bytes_lh_g);
        hipMalloc(&d_LH_conn, bytes_lh_conn);
        hipMalloc(&d_nc_lh_flag, bytes_nc_int);
        hipMalloc(&d_beta_lh, bytes_LH);
        hipMalloc(&d_r_lh, bytes_LH3);
        hipMalloc(&d_LH_q, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hh, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hc, bytes_lh_d);
        hipMalloc(&d_LH_vdw_hl, bytes_lh_d);
        hipMalloc(&d_LH_vdw_ht, bytes_lh_d);
        hipMalloc(&d_LH_kstr, bytes_lh_d);
        hipMalloc(&d_LH_kben, bytes_lh_d);
        hipMalloc(&d_LH_streq, bytes_lh_d);
        hipMalloc(&d_LH_betaeq, bytes_lh_d);
        hipMalloc(&d_LH_force, bytes_LH3);

	hipMalloc(&d_force_m, bytes_r_t_lh_m);
	hipMalloc(&d_torque_m, bytes_r_t_lh_m);
	hipMalloc(&d_t_force_m, bytes_r_t_lh_m);
	hipMalloc(&d_LH_force_m, bytes_r_t_lh_m);
	hipMalloc(&d_temp_force_m, bytes_r_t_lh_m);

	//Copy data to the device

	hipMemcpy(d_Energy, Energy, bytes,hipMemcpyHostToDevice);
	hipMemcpy(d_type, type, bytes_ni, hipMemcpyHostToDevice);
	hipMemcpy(d_r, r, bytes_n3, hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, bytes_n3, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, bytes_n3, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, bytes_n3, hipMemcpyHostToDevice);
	hipMemcpy(d_alpha, alpha, bytes_nd, hipMemcpyHostToDevice);
	hipMemcpy(d_beta, beta, bytes_nd, hipMemcpyHostToDevice);
	hipMemcpy(d_gamma, gamma, bytes_nd, hipMemcpyHostToDevice);
	hipMemcpy(d_length, length, bytes_nd, hipMemcpyHostToDevice);
	hipMemcpy(d_a_dna, a_dna, bytes_nc3, hipMemcpyHostToDevice);
	hipMemcpy(d_b_dna, b_dna, bytes_nc3, hipMemcpyHostToDevice);
	hipMemcpy(d_c_dna, c_dna, bytes_nc3, hipMemcpyHostToDevice);
	hipMemcpy(d_alpha_p, alpha_p, bytes_nc, hipMemcpyHostToDevice);
	hipMemcpy(d_beta_p, beta_p, bytes_nc, hipMemcpyHostToDevice);
	hipMemcpy(d_gamma_p, gamma_p, bytes_nc, hipMemcpyHostToDevice);
	hipMemcpy(d_phi_o, phi_o, bytes_nd, hipMemcpyHostToDevice);
	hipMemcpy(d_core_pos, core_pos, bytes_Nq3, hipMemcpyHostToDevice);
	hipMemcpy(d_core_q, core_q, bytes_Nq, hipMemcpyHostToDevice);
	hipMemcpy(d_force, force, bytes_n3, hipMemcpyHostToDevice);
	hipMemcpy(d_torque, torque, bytes_n3, hipMemcpyHostToDevice);

	hipMemcpy(d_tail_pos, tail_pos, bytes_t3, hipMemcpyHostToDevice);
	hipMemcpy(d_tail_fix, tail_fix, bytes_t, hipMemcpyHostToDevice);
	hipMemcpy(d_nc_t_flag, nc_t_flag, bytes_nc_int, hipMemcpyHostToDevice);
	hipMemcpy(d_r_t, r_t, bytes_tail3, hipMemcpyHostToDevice);
	hipMemcpy(d_beta_t, beta_t, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_h_t, h_t, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_g_t, g_t, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_lo_t, lo_t, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_beta_o_t, beta_o_t, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_t_q, t_q, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_t_rad, t_rad, bytes_tail, hipMemcpyHostToDevice);
	hipMemcpy(d_t_grp, t_grp, bytes_tail_int, hipMemcpyHostToDevice);
	hipMemcpy(d_t_fix, t_fix, bytes_tail_int, hipMemcpyHostToDevice);
	hipMemcpy(d_t_force, t_force, bytes_tail3, hipMemcpyHostToDevice);

	hipMemcpy(d_LH_g_pos, LH_g_pos, bytes_lh_g, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_conn, LH_conn, bytes_lh_conn, hipMemcpyHostToDevice);
        hipMemcpy(d_nc_lh_flag, nc_lh_flag, bytes_nc_int, hipMemcpyHostToDevice);
        hipMemcpy(d_beta_lh, beta_lh, bytes_LH, hipMemcpyHostToDevice);
        hipMemcpy(d_r_lh, r_lh, bytes_LH3, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_q, LH_q, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hh, LH_vdw_hh, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hc, LH_vdw_hc, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_hl, LH_vdw_hl, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_vdw_ht, LH_vdw_ht, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_kstr, LH_kstr, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_kben, LH_kben, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_streq, LH_streq, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_betaeq, LH_betaeq, bytes_lh_d, hipMemcpyHostToDevice);
        hipMemcpy(d_LH_force, LH_force, bytes_LH3, hipMemcpyHostToDevice);


	int threads_per_block_rtlh = 16;
        dim3 block_size_rtlh(threads_per_block_rtlh, threads_per_block_rtlh);
        dim3 grid_size_rtlh(maxrtlh*3 / block_size_rtlh.x + 1 , maxrtlh*3 / block_size_rtlh.y + 1 );

	init_force_torque_m<<<grid_size_rtlh, block_size_rtlh>>>(maxrtlh, d_force_m, d_torque_m, d_t_force_m, d_LH_force_m, d_temp_force_m);


	int num_thread = 64;
        int num_block = (n + num_thread -1) / num_thread;

	mechanical_force_and_torque_cuda <<<num_block, num_thread>>> (maxrtlh, n_c, nc3, n, n3, d_type, d_r, d_a, d_b, d_c, d_alpha, d_beta, d_gamma, d_length, d_a_dna, d_b_dna, d_c_dna, d_alpha_p, d_beta_p, d_gamma_p, h, g, s, d_phi_o, d_force, d_torque, d_Energy, d_force_m, d_torque_m);
	
	//assign number of blocks and threads

        int threads_per_block = 16;
        dim3 block_size(threads_per_block, threads_per_block);
        dim3 grid_size(n / block_size.x + 1 , n / block_size.y + 1 );

	Electrostatic_and_Excluded_volume_force_cuda <<<grid_size, block_size>>> (maxrtlh, n, n3, n_c, nc3, d_type, d_r, d_a, d_b, d_c, debyell, debye, q_l, k_e, k_ex, k_h1, sigma_DNA_DNA, sigma_DNA_Core, sigma_Core_Core, Nq, Nq3, d_core_pos, d_core_q, d_force, d_torque, d_Energy, d_force_m, d_torque_m);
	
	//assign number of blocks and threads

	if (n_tail >0){
		int num_thread_t = 64;
        	int num_block_t = (n_tail + num_thread_t -1) / num_thread_t;

		tail_force_cuda_str_ben<<<num_block_t, num_thread_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t, d_beta_t, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force, d_force, d_torque, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m);

		int threads_per_block_t = 16;
        	dim3 block_size_t(threads_per_block_t, threads_per_block_t);
        	dim3 grid_size_t(n_tail / block_size_t.x + 1 , n_tail / block_size_t.y + 1 );

		tail_force_cuda_ele_vdw<<<grid_size_t, block_size_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t, d_beta_t, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force, d_force, d_torque, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m, d_force_m, d_torque_m);

	}

	//assign number of blocks and threads

	if (n_LH>0){
        	int num_thread_lh = 64;
        	int num_block_lh = (n_LH + num_thread_lh -1) / num_thread_lh;


		Linker_Histone_str_ben_cuda<<<num_block_lh, num_thread_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh, d_r_lh, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t, d_t_q, d_t_force, d_LH_force, d_force, d_torque, d_Energy, d_LH_force_m);

		int threads_per_block_lh = 16;
        	dim3 block_size_lh(threads_per_block_lh, threads_per_block_lh);
        	dim3 grid_size_lh(maxrtlh / block_size_lh.x + 1 , maxrtlh / block_size_lh.y + 1 );

		Linker_Histone_ele_vdw_cuda<<<grid_size_lh, block_size_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh, d_r_lh, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t, d_t_q, d_t_force, d_LH_force, d_force, d_torque, d_Energy, d_LH_force_m, d_force_m, d_torque_m, d_t_force_m, d_temp_force_m);

	}

	int num_thread_rtlh = 64;
        int num_block_rtlh = (maxrtlh + num_thread_rtlh -1) / num_thread_rtlh;

	temp_force_2_force_m <<<num_block_rtlh, num_thread_rtlh>>> (n, maxrtlh, d_t_grp, d_type, d_temp_force_m, d_force_m, d_torque_m);
	force_reduction <<<num_block_rtlh, num_thread_rtlh>>> (n, n_tail, n_LH, maxrtlh, maxrtlh*3, d_force_m, d_force, d_torque_m, d_torque, d_t_force_m, d_t_force, d_LH_force_m, d_LH_force);

	hipMemcpy(force, d_force, bytes_n3, hipMemcpyDeviceToHost);
	hipMemcpy(torque, d_torque, bytes_n3, hipMemcpyDeviceToHost);
	hipMemcpy(t_force, d_t_force, bytes_tail3, hipMemcpyDeviceToHost);
	hipMemcpy(LH_force, d_LH_force, bytes_LH3, hipMemcpyDeviceToHost);
	hipMemcpy(Energy, d_Energy, bytes, hipMemcpyDeviceToHost);

	hipFree(d_Energy);
	hipFree(d_type);
	hipFree(d_r);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(d_gamma);
	hipFree(d_length);
	hipFree(d_a_dna);
	hipFree(d_b_dna);
	hipFree(d_c_dna);
	hipFree(d_alpha_p);
	hipFree(d_beta_p);
	hipFree(d_gamma_p);
	hipFree(d_phi_o);
	hipFree(d_force);
	hipFree(d_torque);
	hipFree(d_core_pos);
        hipFree(d_core_q);

	hipFree(d_tail_pos);
	hipFree(d_tail_fix);
	hipFree(d_nc_t_flag);
	hipFree(d_r_t);
	hipFree(d_beta_t);
	hipFree(d_h_t);
	hipFree(d_g_t);
	hipFree(d_lo_t);
	hipFree(d_beta_o_t);
	hipFree(d_t_q);
	hipFree(d_t_rad);
	hipFree(d_t_grp);
	hipFree(d_t_fix);
	hipFree(d_t_force);

	hipFree(d_LH_g_pos);
        hipFree(d_LH_conn);
        hipFree(d_nc_lh_flag);
        hipFree(d_beta_lh);
        hipFree(d_r_lh);
        hipFree(d_LH_q);
        hipFree(d_LH_vdw_hh);
        hipFree(d_LH_vdw_hc);
        hipFree(d_LH_vdw_hl);
        hipFree(d_LH_vdw_ht);
        hipFree(d_LH_kstr);
        hipFree(d_LH_kben);
        hipFree(d_LH_streq);
        hipFree(d_LH_betaeq);
        hipFree(d_LH_force);

	hipFree(d_force_m);
	hipFree(d_torque_m);
	hipFree(d_t_force_m);
	hipFree(d_LH_force_m);
	hipFree(d_temp_force_m);
}
*/

extern "C++" void main_cuda(int n_c, int nc3, int step, int number_of_steps, double time_step, double del, int frequency_RP, int frequency_of_sampling, double h, double g, double s, double debyell, double debye, double q_l, double k_e, double k_ex, double k_h1, double sigma_DNA_DNA, double sigma_DNA_Core, double sigma_Core_Core, double sigma_Tail_Tail, double sigma_Tail_Linker, double sigma_Tail_Core, int Nq, int Nq3, int n_t, int n_lh_n, int n_lh_g, int n_lh_c, int n, int n3, int n_tail, int n_tail3, int n_LH, int n_LH3, double a1, double a2, double s2dt, double* rr, double* p, double* Energy, double* h_r, double* h_a, double* h_b, double* h_c, double* h_r_t, double* h_r_lh, double* h_rad_all, int* nc_lh_flag){

	hipFree(d_r_lh);
	hipFree(d_r_lh_n);
	hipFree(d_beta_lh);
	hipFree(d_beta_lh_n);
	hipFree(d_LH_force);
	hipFree(d_LH_force_n);
	hipFree(d_r_all);
	hipFree(d_rad_all);
	hipFree(d_LH_force_tmp);
	hipFree(d_nc_lh_flag);

	int maxrtlh;
	int n_D, n_D3;
	n_D = n + n_tail + n_LH;
	n_D3 = n_D*3;

        if (n>n_LH){
                if (n>n_tail){
                        maxrtlh=n;
                }else{
                        maxrtlh=n_tail;
                }
        }else{
                if (n_LH>n_tail){
                        maxrtlh=n_LH;
                }else{
                        maxrtlh=n_tail;
                }
        }

	size_t bytes = sizeof(double);

	size_t bytes_LH = n_LH*sizeof(double);
        size_t bytes_LH3 = n_LH3*sizeof(double);

        size_t bytes_n_D = n_D*sizeof(double);
        size_t bytes_n_D3 = n_D3*sizeof(double);
	size_t bytes_nc_int = n_c*sizeof(int);

	hipMalloc(&d_LH_force, bytes_LH3);
        hipMalloc(&d_LH_force_n, bytes_LH3);

        hipMalloc(&d_LH_force_tmp, bytes_LH3);

        hipMalloc(&d_beta_lh_n, bytes_LH);
        hipMalloc(&d_r_lh_n, bytes_LH3);

        hipMalloc(&d_r_all, bytes_n_D3);
        hipMalloc(&d_rad_all, bytes_n_D);
        hipMalloc(&d_r_lh, bytes_LH3);
	hipMalloc(&d_beta_lh, bytes_LH);

	hipMalloc(&d_nc_lh_flag, bytes_nc_int);

	hipMemcpy(d_r_lh, h_r_lh, bytes_LH3, hipMemcpyHostToDevice);
	hipMemcpy(d_rad_all, h_rad_all, bytes_n_D, hipMemcpyHostToDevice);
	hipMemcpy(d_nc_lh_flag, nc_lh_flag, bytes_nc_int, hipMemcpyHostToDevice);

	int num_thread_lh = 64;
        int num_block_lh = (n_LH + num_thread_lh -1) / num_thread_lh;

	update_LH_beta_cuda <<<num_block_lh, num_thread_lh>>> (n_LH, n_lh_c, d_r_lh, d_beta_lh);

	int num_thread_rtlh3 = 64;
        int num_block_rtlh3 = (maxrtlh*3 + num_thread_rtlh3 -1) / num_thread_rtlh3;

	step_init<<<num_block_rtlh3, num_thread_rtlh3>>>(n, n_tail, n_LH, d_force, d_force_n, d_torque, d_torque_n, d_t_force, d_t_force_n, d_LH_force, d_LH_force_n, d_Energy, d_r_all, d_r, d_r_t, d_r_lh);	

//	if (step%frequency_RP == 0){
	hipFree(d_D);
        hipFree(d_Chol);

	size_t bytes_D = n_D3*n_D3*sizeof(double);
	
	hipMalloc(&d_D, bytes_D);
        hipMalloc(&d_Chol, bytes_D);	

	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);

	int work_size = 0;
	int *devInfo;
	hipMalloc(&devInfo, sizeof(int));

	int threads_per_block_D = 32;
        dim3 block_size_D(threads_per_block_D, threads_per_block_D);
	dim3 grid_size_D(n_D3 / block_size_D.x + 1 , n_D3 / block_size_D.y + 1 );

	Diffusion_Tensor_CUDA <<<grid_size_D, block_size_D>>> (n_D, n_D3, d_r_all, a1, a2, d_rad_all, d_D, d_Chol);

	hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_UPPER, n_D3, d_Chol, n_D3, &work_size);
	double *work;
	hipMalloc(&work, work_size * sizeof(double));
        hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, n_D3, d_Chol, n_D3, work, work_size, devInfo);
	Cholesky_Decomposition_mod <<<grid_size_D, block_size_D>>> (n_D3, d_Chol);
	hipFree(devInfo);
	hipFree(work);

        hipsolverDnDestroy(solver_handle);
//        }

	double* d_p;
	double* d_rr;

	size_t bytes_p = n_D3*sizeof(double);
	size_t bytes_rr = n3*sizeof(double);

	hipMalloc(&d_p, bytes_p);
	hipMalloc(&d_rr, bytes_rr);

	hipMemcpy(d_p, p, bytes_p, hipMemcpyHostToDevice);
	hipMemcpy(d_rr, rr, bytes_rr, hipMemcpyHostToDevice);

	int num_thread_rd = 32;
        int num_block_rd = (n_D3 + num_thread_rd -1) / num_thread_rd;

	rd_cal <<<num_block_rd, num_thread_rd>>> (n_D3, d_rd, d_Chol, d_p, s2dt);

	//Force and torque calculation

	int threads_per_block_rtlh = 16;
        dim3 block_size_rtlh(threads_per_block_rtlh, threads_per_block_rtlh);
        dim3 grid_size_rtlh(maxrtlh*3 / block_size_rtlh.x + 1 , maxrtlh*3 / block_size_rtlh.y + 1 );

        init_force_torque_m<<<grid_size_rtlh, block_size_rtlh>>>(maxrtlh, d_force_m, d_torque_m, d_t_force_m, d_LH_force_m, d_temp_force_m, d_Energy_m);


        int num_thread = 64;
        int num_block = (n + num_thread -1) / num_thread;

        mechanical_force_and_torque_cuda <<<num_block, num_thread>>> (maxrtlh, n_c, nc3, n, n3, d_type, d_r, d_a, d_b, d_c, d_alpha, d_beta, d_gamma, d_length, d_a_dna, d_b_dna, d_c_dna, d_alpha_p, d_beta_p, d_gamma_p, h, g, s, d_phi_o, d_force, d_torque, d_Energy, d_force_m, d_torque_m, d_Energy_m);

        int threads_per_block = 16;
        dim3 block_size(threads_per_block, threads_per_block);
        dim3 grid_size(n / block_size.x + 1 , n / block_size.y + 1 );

        Electrostatic_and_Excluded_volume_force_cuda <<<grid_size, block_size>>> (maxrtlh, n, n3, n_c, nc3, d_type, d_r, d_a, d_b, d_c, debyell, debye, q_l, k_e, k_ex, k_h1, sigma_DNA_DNA, sigma_DNA_Core, sigma_Core_Core, Nq, Nq3, d_core_pos, d_core_q, d_force, d_torque, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	int num_thread_t = 64;
        int num_block_t = (n_tail + num_thread_t -1) / num_thread_t;

	int threads_per_block_t = 16;
        dim3 block_size_t(threads_per_block_t, threads_per_block_t);
        dim3 grid_size_t(n_tail / block_size_t.x + 1 , n_tail / block_size_t.y + 1 );

        if (n_tail >0){

                tail_force_cuda_str_ben<<<num_block_t, num_thread_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t, d_beta_t, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force, d_force, d_torque, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m, d_Energy_m);


                tail_force_cuda_ele_vdw<<<grid_size_t, block_size_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t, d_beta_t, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force, d_force, d_torque, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m, d_force_m, d_torque_m, d_Energy_m);

        }

//	int num_thread_lh = 64;
//        int num_block_lh = (n_LH + num_thread_lh -1) / num_thread_lh;

	int threads_per_block_lh = 16;
        dim3 block_size_lh(threads_per_block_lh, threads_per_block_lh);
        dim3 grid_size_lh(maxrtlh / block_size_lh.x + 1 , maxrtlh / block_size_lh.y + 1 );

        if (n_LH>0){


                Linker_Histone_str_ben_cuda<<<num_block_lh, num_thread_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh, d_r_lh, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t, d_t_q, d_t_force, d_LH_force, d_force, d_torque, d_Energy, d_LH_force_m, d_Energy_m);


                Linker_Histone_ele_vdw_cuda<<<grid_size_lh, block_size_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh, d_r_lh, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r, d_a, d_b, d_c, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t, d_t_q, d_t_force, d_LH_force, d_force, d_torque, d_Energy, d_LH_force_m, d_force_m, d_torque_m, d_t_force_m, d_temp_force_m, d_Energy_m);

        }

        int num_thread_rtlh = 64;
        int num_block_rtlh = (maxrtlh + num_thread_rtlh -1) / num_thread_rtlh;

        temp_force_2_force_m <<<num_block_rtlh, num_thread_rtlh>>> (n, maxrtlh, d_t_grp, d_type, d_temp_force_m, d_force_m, d_torque_m);
        force_reduction <<<num_block_rtlh, num_thread_rtlh>>> (n, n_tail, n_LH, maxrtlh, maxrtlh*3, d_force_m, d_force, d_torque_m, d_torque, d_t_force_m, d_t_force, d_LH_force_m, d_LH_force);

	Energy_reduction  <<<num_block_rtlh, num_thread_rtlh>>> (maxrtlh, d_Energy, d_Energy_m);

	hipMemcpy(Energy, d_Energy, bytes, hipMemcpyDeviceToHost);

	rotation_cal <<<num_block, num_thread>>> (n, d_d_theta, d_type, time_step, d_torque, d_rr, d_a, d_b, d_c, d_a_n, d_b_n, d_c_n);

	translation_cal <<<num_block_rd, num_thread_rd>>> (n_D3, n3, n_tail3, n_LH3, d_r, d_r_n, d_rd, d_r_t, d_r_t_n, d_r_lh, d_r_lh_n, del, d_force, d_t_force, d_LH_force, d_D);

	update_tail_beta_cuda <<<num_block_t, num_thread_t>>> (n_tail, d_r_t_n, d_beta_t_n, d_t_grp);

	update_LH_beta_cuda <<<num_block_lh, num_thread_lh>>> (n_LH, n_lh_c, d_r_lh_n, d_beta_lh_n);
	
	update_Euler_Angle_cuda <<<num_block, num_thread>>> (n_c, nc3, n, n3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, d_alpha_n, d_beta_n, d_gamma_n, d_length_n, d_a_dna_n, d_b_dna_n, d_c_dna_n, d_alpha_p_n, d_beta_p_n, d_gamma_p_n);

	
	init_force_torque_m<<<grid_size_rtlh, block_size_rtlh>>>(maxrtlh, d_force_m, d_torque_m, d_t_force_m, d_LH_force_m, d_temp_force_m, d_Energy_m);

	mechanical_force_and_torque_cuda <<<num_block, num_thread>>> (maxrtlh, n_c, nc3, n, n3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, d_alpha_n, d_beta_n, d_gamma_n, d_length_n, d_a_dna_n, d_b_dna_n, d_c_dna_n, d_alpha_p_n, d_beta_p_n, d_gamma_p_n, h, g, s, d_phi_o, d_force_n, d_torque_n, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	Electrostatic_and_Excluded_volume_force_cuda <<<grid_size, block_size>>> (maxrtlh, n, n3, n_c, nc3, d_type, d_r_n, d_a_n, d_b_n, d_c_n, debyell, debye, q_l, k_e, k_ex, k_h1, sigma_DNA_DNA, sigma_DNA_Core, sigma_Core_Core, Nq, Nq3, d_core_pos, d_core_q, d_force_n, d_torque_n, d_Energy, d_force_m, d_torque_m, d_Energy_m);

	if (n_tail >0){
		tail_force_cuda_str_ben<<<num_block_t, num_thread_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t_n, d_beta_t_n, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r_n, d_a_n, d_b_n, d_c_n, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force_n, d_force_n, d_torque_n, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m, d_Energy_m);
		tail_force_cuda_ele_vdw<<<grid_size_t, block_size_t>>>(maxrtlh, n, n_t, d_type, n_tail, n_tail3, d_tail_pos, d_tail_fix, d_nc_t_flag, d_r_t_n, d_beta_t_n, d_h_t, d_g_t, d_lo_t, d_beta_o_t, h, k_e, debye, k_ex, d_t_q, d_t_rad, d_t_grp, d_t_fix, d_r_n, d_a_n, d_b_n, d_c_n, q_l, Nq, Nq3, d_core_pos, d_core_q, d_t_force_n, d_force_n, d_torque_n, sigma_Tail_Tail, sigma_Tail_Linker, sigma_Tail_Core, d_Energy, d_t_force_m, d_force_m, d_torque_m, d_Energy_m);

        }

	if (n_LH>0){
		Linker_Histone_str_ben_cuda<<<num_block_lh, num_thread_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh_n, d_r_lh_n, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r_n, d_a_n, d_b_n, d_c_n, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t_n, d_t_q, d_t_force_n, d_LH_force_n, d_force_n, d_torque_n, d_Energy, d_LH_force_m, d_Energy_m);
		Linker_Histone_ele_vdw_cuda<<<grid_size_lh, block_size_lh>>>(maxrtlh, n, n_tail, n_lh_n, n_lh_g, n_lh_c, d_t_grp, d_t_fix, d_type, n_LH, n_LH3, d_LH_g_pos, d_LH_conn, d_nc_lh_flag, d_beta_lh_n, d_r_lh_n, d_LH_q, k_e, debye, k_ex, d_LH_vdw_hh, d_LH_vdw_hc, d_LH_vdw_hl, d_LH_vdw_ht, d_LH_kstr, d_LH_kben, d_LH_streq, d_LH_betaeq, d_r_n, d_a_n, d_b_n, d_c_n, q_l, Nq, Nq3, d_core_pos, d_core_q, d_r_t_n, d_t_q, d_t_force_n, d_LH_force_n, d_force_n, d_torque_n, d_Energy, d_LH_force_m, d_force_m, d_torque_m, d_t_force_m, d_temp_force_m, d_Energy_m);
        }

	temp_force_2_force_m <<<num_block_rtlh, num_thread_rtlh>>> (n, maxrtlh, d_t_grp, d_type, d_temp_force_m, d_force_m, d_torque_m);
        force_reduction <<<num_block_rtlh, num_thread_rtlh>>> (n, n_tail, n_LH, maxrtlh, maxrtlh*3, d_force_m, d_force_n, d_torque_m, d_torque_n, d_t_force_m, d_t_force_n, d_LH_force_m, d_LH_force_n);

	force_torque_tmp<<<num_block_rtlh3, num_thread_rtlh3>>>(n3, n_tail3, n_LH3, d_force, d_torque, d_t_force, d_LH_force, d_force_n, d_torque_n, d_t_force_n, d_LH_force_n, d_force_tmp, d_torque_tmp, d_t_force_tmp, d_LH_force_tmp);

	rotation_cal <<<num_block, num_thread>>> (n, d_d_theta, d_type, time_step, d_torque_tmp, d_rr, d_a, d_b, d_c, d_a_n, d_b_n, d_c_n);

        translation_cal <<<num_block_rd, num_thread_rd>>> (n_D3, n3, n_tail3, n_LH3, d_r, d_r_n, d_rd, d_r_t, d_r_t_n, d_r_lh, d_r_lh_n, del, d_force_tmp, d_t_force_tmp, d_LH_force_tmp, d_D);

	final_updates <<<num_block_rtlh3, num_thread_rtlh3>>> (n3, n_tail3, n_LH3, d_r, d_r_n, d_a, d_a_n, d_b, d_b_n, d_c, d_c_n, d_r_t, d_r_t_n, d_r_lh, d_r_lh_n);

	update_Euler_Angle_cuda <<<num_block, num_thread>>> (n_c, nc3, n, n3, d_type, d_r, d_a, d_b, d_c, d_alpha, d_beta, d_gamma, d_length, d_a_dna, d_b_dna, d_c_dna, d_alpha_p, d_beta_p, d_gamma_p);

	update_tail_beta_cuda <<<num_block_t, num_thread_t>>> (n_tail, d_r_t, d_beta_t, d_t_grp);

	update_LH_beta_cuda <<<num_block_lh, num_thread_lh>>> (n_LH, n_lh_c, d_r_lh, d_beta_lh);

	size_t bytes_r_lh = n_LH3*sizeof(double);
	hipMemcpy(h_r_lh, d_r_lh, bytes_r_lh, hipMemcpyDeviceToHost);

	if (step%frequency_of_sampling == 0 or step == number_of_steps-1){
		size_t bytes_r = n3*sizeof(double);		
		size_t bytes_r_t = n_tail3*sizeof(double);

		hipMemcpy(h_r, d_r, bytes_r, hipMemcpyDeviceToHost);
		hipMemcpy(h_a, d_a, bytes_r, hipMemcpyDeviceToHost);
		hipMemcpy(h_b, d_b, bytes_r, hipMemcpyDeviceToHost);
		hipMemcpy(h_c, d_c, bytes_r, hipMemcpyDeviceToHost);
		hipMemcpy(h_r_t, d_r_t, bytes_r_t, hipMemcpyDeviceToHost);
	}



	hipFree(d_p);
        hipFree(d_rr);
}

extern "C++" void free_all(){

	hipFree(d_Energy);
        hipFree(d_type);
        hipFree(d_r);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipFree(d_alpha);
        hipFree(d_beta);
        hipFree(d_gamma);
        hipFree(d_length);
        hipFree(d_a_dna);
        hipFree(d_b_dna);
        hipFree(d_c_dna);
        hipFree(d_alpha_p);
        hipFree(d_beta_p);
        hipFree(d_gamma_p);
        hipFree(d_phi_o);
        hipFree(d_force);
        hipFree(d_torque);
        hipFree(d_core_pos);
        hipFree(d_core_q);

	hipFree(d_r_n);
        hipFree(d_a_n);
        hipFree(d_b_n);
        hipFree(d_c_n);
        hipFree(d_alpha_n);
        hipFree(d_beta_n);
        hipFree(d_gamma_n);
        hipFree(d_length_n);
        hipFree(d_a_dna_n);
        hipFree(d_b_dna_n);
        hipFree(d_c_dna_n);
        hipFree(d_alpha_p_n);
        hipFree(d_beta_p_n);
        hipFree(d_gamma_p_n);

	hipFree(d_force_n);
        hipFree(d_torque_n);
	hipFree(d_force_tmp);
        hipFree(d_torque_tmp);

	hipFree(d_r_all);
	hipFree(d_rad_all);
	hipFree(d_d_theta);
	hipFree(d_rd);

        hipFree(d_tail_pos);
        hipFree(d_tail_fix);
        hipFree(d_nc_t_flag);
        hipFree(d_r_t);
        hipFree(d_beta_t);
        hipFree(d_h_t);
        hipFree(d_g_t);
        hipFree(d_lo_t);
        hipFree(d_beta_o_t);
        hipFree(d_t_q);
        hipFree(d_t_rad);
        hipFree(d_t_grp);
        hipFree(d_t_fix);
        hipFree(d_t_force);

	hipFree(d_r_t_n);
        hipFree(d_beta_t_n);

	hipFree(d_t_force_n);
	hipFree(d_t_force_tmp);

        hipFree(d_LH_g_pos);
        hipFree(d_LH_conn);
        hipFree(d_nc_lh_flag);
        hipFree(d_beta_lh);
        hipFree(d_r_lh);
        hipFree(d_LH_q);
        hipFree(d_LH_vdw_hh);
        hipFree(d_LH_vdw_hc);
        hipFree(d_LH_vdw_hl);
        hipFree(d_LH_vdw_ht);
        hipFree(d_LH_kstr);
        hipFree(d_LH_kben);
        hipFree(d_LH_streq);
        hipFree(d_LH_betaeq);
        hipFree(d_LH_force);

	hipFree(d_beta_lh_n);
        hipFree(d_r_lh_n);

	hipFree(d_LH_force_n);
	hipFree(d_LH_force_tmp);

        hipFree(d_force_m);
        hipFree(d_torque_m);
        hipFree(d_t_force_m);
        hipFree(d_LH_force_m);
        hipFree(d_temp_force_m);

	hipFree(d_D);
	hipFree(d_Chol);

	hipFree(d_Energy_m);
}
